#include "hip/hip_runtime.h"

#include "kernel_regression_cuda.h"
#include "auxilliary_functions.hpp"
#include "Rodeo_macros.hpp"
#include "test_functions.hpp"

#include<stdio.h>
#include<iostream>
#include<math.h>



#include <armadillo>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <codi.hpp>

using namespace arma;


//This implementation using CAS incurs a non-trivial cost though.
//Had to use this because compute < 600 doesn't support atomic add with float and > 600 throws up some MemCpy - invalid code error
//__device__ float atomicDAdd(float* address, float val);
//
//__device__ float atomicDAdd(float* address, float val)
//{
//	unsigned long long int* address_as_ull =
//			(unsigned long long int*)address;
//	unsigned long long int old = *address_as_ull, assumed;
//	do {
//		assumed = old;
//		old = atomicCAS(address_as_ull, assumed,
//				__float_as_longlong(val +
//						__longlong_as_float(assumed)));
//	} while (assumed != old);
//	return __longlong_as_float(old);
//}


//__managed__ float MDevice[numVar*numVar+1];
__constant__ float MDevice[numVar*numVar+1];


float gaussianKernel(frowvec &xi, frowvec &xj, float sigma, fmat &M) {
#if 0
	printf("calling gaussianKernel...\n");
	xi.print();
	xj.print();
#endif

	/* calculate distance between xi and xj with the matrix M */
	float metricVal = calcMetric(xi, xj, M);
#if 0
	printf("metricVal = %10.7f\n",metricVal);
#endif

	float sqr_two_pi = sqrt(2.0 * datum::pi);

	float kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));

	kernelVal += 10E-14;

#if 0
	printf("kernelVal = %10.7f\n",kernelVal);

#endif
	return kernelVal;

}


double gaussianKernel(rowvec &xi, rowvec &xj, double sigma, mat &M) {

	/* calculate distance between xi and xj with the matrix M */
	double metricVal = calcMetric(xi, xj, M);

	double sqr_two_pi = sqrt(2.0 * datum::pi);

	double kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));

	kernelVal += 10E-14;

	return kernelVal;

}




float SIGN(float a, float b) {

	if (b >= 0.0) {
		return fabs(a);
	} else {
		return -fabs(a);
	}
}

codi::RealReverse SIGN(codi::RealReverse a, codi::RealReverse b) {

	if (b >= 0.0) {
		return fabs(a);
	} else {

		return -fabs(a);
	}
}

codi::RealForward SIGN(codi::RealForward a, codi::RealForward b) {

	if (b >= 0.0) {
		return fabs(a);
	} else {

		return -fabs(a);
	}
}

float PYTHAG(float a, float b) {
	float at = fabs(a), bt = fabs(b), ct, result;

	if (at > bt) {
		ct = bt / at;
		result = at * sqrt(1.0 + ct * ct);
	} else if (bt > 0.0) {
		ct = at / bt;
		result = bt * sqrt(1.0 + ct * ct);
	} else
		result = 0.0;
	return (result);
}

codi::RealReverse PYTHAG(codi::RealReverse a, codi::RealReverse b) {
	codi::RealReverse at = fabs(a), bt = fabs(b), ct, result;

	if (at > bt) {
		ct = bt / at;
		result = at * sqrt(1.0 + ct * ct);
	} else if (bt > 0.0) {
		ct = at / bt;
		result = bt * sqrt(1.0 + ct * ct);
	} else
		result = 0.0;
	return (result);
}
codi::RealForward PYTHAG(codi::RealForward a, codi::RealForward b) {
	codi::RealForward at = fabs(a), bt = fabs(b), ct, result;

	if (at > bt) {
		ct = bt / at;
		result = at * sqrt(1.0 + ct * ct);
	} else if (bt > 0.0) {
		ct = at / bt;
		result = bt * sqrt(1.0 + ct * ct);
	} else
		result = 0.0;
	return (result);
}

/** calculate regularization terms for the given matrix L
 *
 * @param[in]  L:  lower diagonal matrix
 * @param[in]  wSvd: weight for the svd regularization part
 * @param[in]  w12: weight for the mixed 12 regularization part 
 * @param[out] regTerm 
 *
 */

int calcRegTerms(float *L, float *regTerm, float wSvd, float w12, int dim) {
	int flag, i, its, j, jj, k, l = 0, nm;
	float c, f, h, s, x, y, z;
	float anorm = 0.0, g = 0.0, scale = 0.0;


	int m = dim;
	int n = dim;


	float **a;
	a = new float*[dim];

	for (i = 0; i < dim; i++) {

		a[i] = new float[dim];
	}

	float **M;
	M= new float*[dim];

	for (i = 0; i < dim; i++) {

		M[i] = new float[dim];
	}


	float **LT;
	LT = new float*[dim];
	for (int i = 0; i < dim; i++) {
		LT[i] = new float[dim];

	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < dim; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = L[i*dim+j];
		}


	}

#if 0
	printf("L = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", L[i*dim+j]);

		}
		printf("\n");
	}	
#endif


#if 0
	printf("LT = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", LT[i][j]);

		}
		printf("\n");
	}

#endif

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=0;
			M[i][j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
			for(int k = 0; k < dim; ++k)
			{
				M[i][j] += L[i*dim+k] * LT[k][j];

			}

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=M[i][j];

		}



#if 0
	printf("a = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", a[i][j]);

		}
		printf("\n");
	}

#endif

#if 0
	/* only for validation */
	mat Lval(dim,dim);
	mat LTval(dim,dim);
	mat aval(dim,dim);
	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {
			Lval(i,j) = Lin(i,j);
		}
	LTval = trans(Lval);
	aval = Lval*LTval;

	printf("aval = \n");
	aval.print();

#endif


	/* SVD part */

	float **v;
	v = new float*[n];

	for (i = 0; i < n; i++) {

		v[i] = new float[n];
	}
	float *w = new float[n];

	float *rv1 = new float[n];

	/* Householder reduction to bidiagonal form */
	for (i = 0; i < n; i++) {
		/* left-hand reduction */
		l = i + 1;
		rv1[i] = scale * g;
		g = s = scale = 0.0;
		if (i < m) {
			for (k = i; k < m; k++)
				scale += fabs(a[k][i]);
			if (scale) {
				for (k = i; k < m; k++) {
					a[k][i] = (a[k][i] / scale);
					s += (a[k][i] * a[k][i]);
				}
				f = a[i][i];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][i] = (f - g);
				if (i != n - 1) {
					for (j = l; j < n; j++) {
						for (s = 0.0, k = i; k < m; k++)
							s += (a[k][i] * a[k][j]);
						f = s / h;
						for (k = i; k < m; k++)
							a[k][j] += (f * a[k][i]);
					}
				}
				for (k = i; k < m; k++)
					a[k][i] = (a[k][i] * scale);
			}
		}
		w[i] = (scale * g);

		/* right-hand reduction */
		g = s = scale = 0.0;
		if (i < m && i != n - 1) {
			for (k = l; k < n; k++)
				scale += fabs(a[i][k]);
			if (scale) {
				for (k = l; k < n; k++) {
					a[i][k] = (a[i][k] / scale);
					s += (a[i][k] * a[i][k]);
				}
				f = a[i][l];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][l] = (f - g);
				for (k = l; k < n; k++)
					rv1[k] = a[i][k] / h;
				if (i != m - 1) {
					for (j = l; j < m; j++) {
						for (s = 0.0, k = l; k < n; k++)
							s += (a[j][k] * a[i][k]);
						for (k = l; k < n; k++)
							a[j][k] += (s * rv1[k]);
					}
				}
				for (k = l; k < n; k++)
					a[i][k] = (a[i][k] * scale);
			}
		}
		anorm = MAX(anorm, (fabs(w[i]) + fabs(rv1[i])));
	}

	/* accumulate the right-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		if (i < n - 1) {
			if (g) {
				for (j = l; j < n; j++)
					v[j][i] = ((a[i][j] / a[i][l]) / g);
				/* float division to avoid underflow */
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < n; k++)
						s += (a[i][k] * v[k][j]);
					for (k = l; k < n; k++)
						v[k][j] += (s * v[k][i]);
				}
			}
			for (j = l; j < n; j++)
				v[i][j] = v[j][i] = 0.0;
		}
		v[i][i] = 1.0;
		g = rv1[i];
		l = i;
	}

	/* accumulate the left-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		l = i + 1;
		g = w[i];
		if (i < n - 1)
			for (j = l; j < n; j++)
				a[i][j] = 0.0;
		if (g) {
			g = 1.0 / g;
			if (i != n - 1) {
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < m; k++)
						s += (a[k][i] * a[k][j]);
					f = (s / a[i][i]) * g;
					for (k = i; k < m; k++)
						a[k][j] += (f * a[k][i]);
				}
			}
			for (j = i; j < m; j++)
				a[j][i] = (a[j][i] * g);
		} else {
			for (j = i; j < m; j++)
				a[j][i] = 0.0;
		}
		++a[i][i];
	}

	/* diagonalize the bidiagonal form */
	for (k = n - 1; k >= 0; k--) { /* loop over singular values */
		for (its = 0; its < 30000; its++) { /* loop over allowed iterations */
			flag = 1;
			for (l = k; l >= 0; l--) { /* test for splitting */
				nm = l - 1;
				if (fabs(rv1[l]) + anorm == anorm) {
					flag = 0;
					break;
				}
				if (fabs(w[nm]) + anorm == anorm)
					break;
			}
			if (flag) {
				c = 0.0;
				s = 1.0;
				for (i = l; i <= k; i++) {
					f = s * rv1[i];
					if (fabs(f) + anorm != anorm) {
						g = w[i];
						h = PYTHAG(f, g);
						w[i] = h;
						h = 1.0 / h;
						c = g * h;
						s = (-f * h);
						for (j = 0; j < m; j++) {
							y = a[j][nm];
							z = a[j][i];
							a[j][nm] = (y * c + z * s);
							a[j][i] = (z * c - y * s);
						}
					}
				}
			}
			z = w[k];
			if (l == k) { /* convergence */
				if (z < 0.0) { /* make singular value nonnegative */
					w[k] = (-z);
					for (j = 0; j < n; j++)
						v[j][k] = (-v[j][k]);
				}
				break;
			}
			if (its >= 30000) {
				delete[] rv1;
				fprintf(stderr, "No convergence after 30,000! iterations \n");
				return 1;
			}

			/* shift from bottom 2 x 2 minor */
			x = w[l];
			nm = k - 1;
			y = w[nm];
			g = rv1[nm];
			h = rv1[k];
			f = ((y - z) * (y + z) + (g - h) * (g + h)) / (2.0 * h * y);
			g = PYTHAG(f, 1.0);
			f = ((x - z) * (x + z) + h * ((y / (f + SIGN(g, f))) - h)) / x;

			/* next QR transformation */
			c = s = 1.0;
			for (j = l; j <= nm; j++) {
				i = j + 1;
				g = rv1[i];
				y = w[i];
				h = s * g;
				g = c * g;
				z = PYTHAG(f, h);
				rv1[j] = z;
				c = f / z;
				s = h / z;
				f = x * c + g * s;
				g = g * c - x * s;
				h = y * s;
				y = y * c;
				for (jj = 0; jj < n; jj++) {
					x = v[jj][j];
					z = v[jj][i];
					v[jj][j] = (x * c + z * s);
					v[jj][i] = (z * c - x * s);
				}
				z = PYTHAG(f, h);
				w[j] = z;
				if (z) {
					z = 1.0 / z;
					c = f * z;
					s = h * z;
				}
				f = (c * g) + (s * y);
				x = (c * y) - (s * g);
				for (jj = 0; jj < m; jj++) {
					y = a[jj][j];
					z = a[jj][i];
					a[jj][j] = (y * c + z * s);
					a[jj][i] = (z * c - y * s);
				}
			}
			rv1[l] = 0.0;
			rv1[k] = f;
			w[k] = x;
		}
	}
	delete[] rv1;

#if 0
	printf("singular values of a=\n");

	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i]);
	}
#endif

	/* sort the singular values */

	float temp;
	for (i = 0; i < n; ++i) {
		for (j = i + 1; j < n; ++j) {

			if (w[i] < w[j])

			{
				temp = w[i];
				w[i] = w[j];
				w[j] = temp;
			}
		}
	}

#if 0
	printf("singular values of a=\n");


	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i]);
	}
#endif

	/* normalization */
	float wsum = 0.0;
	for (i = 0; i < n; i++) {

		wsum += w[i];

	}

	for (i = 0; i < n; i++) {

		w[i] = w[i]/wsum;

	}

#if 0
	printf("singular values of a (normalized) with wsum =%10.7f\n",wsum);


	for (i = 0; i < n; i++) {

		printf("%15.10f\n",w[i]);
	}
#endif


	float svd_multiplier = (1.0*n*(1.0*n+1))/2.0;

	svd_multiplier = 1.0/svd_multiplier;
#if 0
	printf("svd_multiplier = %10.7f\n",svd_multiplier);
#endif
	float reg_term_svd = 0.0;

	for (i = 0; i < n; i++) {
#if 0
		printf("%d * %10.7f = %10.7f\n",i+1,w[i],(i+1)*w[i]);
#endif
		reg_term_svd = reg_term_svd + (i + 1) * w[i];
	}
#if 0
	printf("reg_term_svd = %10.7f\n",reg_term_svd);
#endif


	float reg_term_L1 = 0.0;

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++) {

			reg_term_L1 = reg_term_L1 + M[i][j]* M[i][j];
		}
#if 0
	printf("reg_term_L1 = %10.7f\n",reg_term_L1);
#endif



	for (i = 0; i < n; i++) {
		delete[] v[i];
		delete[] a[i];
		delete[] M[i];
		delete[] LT[i];
	}


	delete[] LT;
	delete[] M;
	delete[] a;
	delete[] v;
	delete[] w;



	*regTerm = wSvd * svd_multiplier *reg_term_svd + w12 * reg_term_L1;
#if 0
	printf("result = %10.7f\n",*regTerm);
#endif

	return 0;




}

/* forward mode */

int calcRegTerms(float *L, float *regTerm,float *regTermd, float wSvd, float w12, int dim, int derIndx) {
	int flag, i, its, j, jj, k, l = 0, nm;
	codi::RealForward c, f, h, s, x, y, z;
	codi::RealForward anorm = 0.0, g = 0.0, scale = 0.0;


	int m = dim;
	int n = dim;

	codi::RealForward *Lcodi = new codi::RealForward[dim*dim];
	for (int i = 0; i < dim*dim; i++) {

		Lcodi[i] = L[i];

	}

	Lcodi[derIndx].setGradient(1.0); 


	codi::RealForward **a;
	a = new codi::RealForward*[dim];

	for (i = 0; i < dim; i++) {

		a[i] = new codi::RealForward[dim];
	}

	codi::RealForward **M;
	M= new codi::RealForward*[dim];

	for (i = 0; i < dim; i++) {

		M[i] = new codi::RealForward[dim];
	}


	codi::RealForward **LT;
	LT = new codi::RealForward*[dim];
	for (int i = 0; i < dim; i++) {
		LT[i] = new codi::RealForward[dim];

	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < dim; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = Lcodi[i*dim+j];
		}


	}
#if 0
	printf("LT = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", LT[i][j]);

		}
		printf("\n");
	}

#endif

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=0;
			M[i][j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
			for(int k = 0; k < dim; ++k)
			{
				M[i][j] += Lcodi[i*dim+k] * LT[k][j];

			}

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=M[i][j];

		}



#if 0
	printf("a = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", a[i][j]);

		}
		printf("\n");
	}

#endif




	/* SVD part */

	codi::RealForward **v;
	v = new codi::RealForward*[n];

	for (i = 0; i < n; i++) {

		v[i] = new codi::RealForward[n];
	}
	codi::RealForward *w = new codi::RealForward[n];

	codi::RealForward *rv1 = new codi::RealForward[n];

	/* Householder reduction to bidiagonal form */
	for (i = 0; i < n; i++) {
		/* left-hand reduction */
		l = i + 1;
		rv1[i] = scale * g;
		g = s = scale = 0.0;
		if (i < m) {
			for (k = i; k < m; k++)
				scale += fabs(a[k][i]);
			if (scale!= 0) {
				for (k = i; k < m; k++) {
					a[k][i] = (a[k][i] / scale);
					s += (a[k][i] * a[k][i]);
				}
				f = a[i][i];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][i] = (f - g);
				if (i != n - 1) {
					for (j = l; j < n; j++) {
						for (s = 0.0, k = i; k < m; k++)
							s += (a[k][i] * a[k][j]);
						f = s / h;
						for (k = i; k < m; k++)
							a[k][j] += (f * a[k][i]);
					}
				}
				for (k = i; k < m; k++)
					a[k][i] = (a[k][i] * scale);
			}
		}
		w[i] = (scale * g);

		/* right-hand reduction */
		g = s = scale = 0.0;
		if (i < m && i != n - 1) {
			for (k = l; k < n; k++)
				scale += fabs(a[i][k]);
			if (scale!=0) {
				for (k = l; k < n; k++) {
					a[i][k] = (a[i][k] / scale);
					s += (a[i][k] * a[i][k]);
				}
				f = a[i][l];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][l] = (f - g);
				for (k = l; k < n; k++)
					rv1[k] = a[i][k] / h;
				if (i != m - 1) {
					for (j = l; j < m; j++) {
						for (s = 0.0, k = l; k < n; k++)
							s += (a[j][k] * a[i][k]);
						for (k = l; k < n; k++)
							a[j][k] += (s * rv1[k]);
					}
				}
				for (k = l; k < n; k++)
					a[i][k] = (a[i][k] * scale);
			}
		}
		anorm = MAX(anorm, (fabs(w[i]) + fabs(rv1[i])));
	}

	/* accumulate the right-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		if (i < n - 1) {
			if (g!=0) {
				for (j = l; j < n; j++)
					v[j][i] = ((a[i][j] / a[i][l]) / g);
				/* float division to avoid underflow */
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < n; k++)
						s += (a[i][k] * v[k][j]);
					for (k = l; k < n; k++)
						v[k][j] += (s * v[k][i]);
				}
			}
			for (j = l; j < n; j++)
				v[i][j] = v[j][i] = 0.0;
		}
		v[i][i] = 1.0;
		g = rv1[i];
		l = i;
	}

	/* accumulate the left-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		l = i + 1;
		g = w[i];
		if (i < n - 1)
			for (j = l; j < n; j++)
				a[i][j] = 0.0;
		if (g!=0) {
			g = 1.0 / g;
			if (i != n - 1) {
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < m; k++)
						s += (a[k][i] * a[k][j]);
					f = (s / a[i][i]) * g;
					for (k = i; k < m; k++)
						a[k][j] += (f * a[k][i]);
				}
			}
			for (j = i; j < m; j++)
				a[j][i] = (a[j][i] * g);
		} else {
			for (j = i; j < m; j++)
				a[j][i] = 0.0;
		}
		++a[i][i];
	}

	/* diagonalize the bidiagonal form */
	for (k = n - 1; k >= 0; k--) { /* loop over singular values */
		for (its = 0; its < 30000; its++) { /* loop over allowed iterations */
			flag = 1;
			for (l = k; l >= 0; l--) { /* test for splitting */
				nm = l - 1;
				if (fabs(rv1[l]) + anorm == anorm) {
					flag = 0;
					break;
				}
				if (fabs(w[nm]) + anorm == anorm)
					break;
			}
			if (flag) {
				c = 0.0;
				s = 1.0;
				for (i = l; i <= k; i++) {
					f = s * rv1[i];
					if (fabs(f) + anorm != anorm) {
						g = w[i];
						h = PYTHAG(f, g);
						w[i] = h;
						h = 1.0 / h;
						c = g * h;
						s = (-f * h);
						for (j = 0; j < m; j++) {
							y = a[j][nm];
							z = a[j][i];
							a[j][nm] = (y * c + z * s);
							a[j][i] = (z * c - y * s);
						}
					}
				}
			}
			z = w[k];
			if (l == k) { /* convergence */
				if (z < 0.0) { /* make singular value nonnegative */
					w[k] = (-z);
					for (j = 0; j < n; j++)
						v[j][k] = (-v[j][k]);
				}
				break;
			}
			if (its >= 30000) {
				delete[] rv1;
				fprintf(stderr, "No convergence after 30,000! iterations \n");
				return 1;
			}

			/* shift from bottom 2 x 2 minor */
			x = w[l];
			nm = k - 1;
			y = w[nm];
			g = rv1[nm];
			h = rv1[k];
			f = ((y - z) * (y + z) + (g - h) * (g + h)) / (2.0 * h * y);
			g = PYTHAG(f, 1.0);
			f = ((x - z) * (x + z) + h * ((y / (f + SIGN(g, f))) - h)) / x;

			/* next QR transformation */
			c = s = 1.0;
			for (j = l; j <= nm; j++) {
				i = j + 1;
				g = rv1[i];
				y = w[i];
				h = s * g;
				g = c * g;
				z = PYTHAG(f, h);
				rv1[j] = z;
				c = f / z;
				s = h / z;
				f = x * c + g * s;
				g = g * c - x * s;
				h = y * s;
				y = y * c;
				for (jj = 0; jj < n; jj++) {
					x = v[jj][j];
					z = v[jj][i];
					v[jj][j] = (x * c + z * s);
					v[jj][i] = (z * c - x * s);
				}
				z = PYTHAG(f, h);
				w[j] = z;
				if (z!=0) {
					z = 1.0 / z;
					c = f * z;
					s = h * z;
				}
				f = (c * g) + (s * y);
				x = (c * y) - (s * g);
				for (jj = 0; jj < m; jj++) {
					y = a[jj][j];
					z = a[jj][i];
					a[jj][j] = (y * c + z * s);
					a[jj][i] = (z * c - y * s);
				}
			}
			rv1[l] = 0.0;
			rv1[k] = f;
			w[k] = x;
		}
	}
	delete[] rv1;

#if 0
	printf("singular values of a=\n");

	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i]);
	}
#endif

	/* sort the singular values */

	codi::RealForward temp;
	for (i = 0; i < n; ++i) {
		for (j = i + 1; j < n; ++j) {

			if (w[i] < w[j])

			{
				temp = w[i];
				w[i] = w[j];
				w[j] = temp;
			}
		}
	}

#if 0
	printf("singular values of a=\n");


	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i].getValue());
	}
#endif

	/* normalization */
	codi::RealForward wsum = 0.0;
	for (i = 0; i < n; i++) {

		wsum += w[i];

	}

	for (i = 0; i < n; i++) {

		w[i] = w[i]/wsum;

	}

#if 0
	printf("singular values of a (normalized) with wsum =%10.7f\n",wsum.getValue());


	for (i = 0; i < n; i++) {

		printf("%15.10f\n",w[i].getValue());
	}
#endif


	float svd_multiplier = (1.0*n*(1.0*n+1))/2.0;

	svd_multiplier = 1.0/svd_multiplier;
#if 0
	printf("svd_multiplier = %10.7f\n",svd_multiplier);
#endif
	codi::RealForward reg_term_svd = 0.0;

	for (i = 0; i < n; i++) {
#if 0
		printf("%d * %10.7f = %10.7f\n",i+1,w[i].getValue(),(i+1)*w[i].getValue());
#endif
		reg_term_svd = reg_term_svd + (i + 1) * w[i];
	}
#if 0
	printf("reg_term_svd = %10.7f\n",reg_term_svd.getValue());
#endif


	codi::RealForward reg_term_L1 = 0.0;

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++) {

			reg_term_L1 = reg_term_L1 + M[i][j]* M[i][j];
		}
#if 0
	printf("reg_term_L1 = %10.7f\n",reg_term_L1.getValue());
#endif



	for (i = 0; i < n; i++) {
		delete[] v[i];
		delete[] a[i];
		delete[] M[i];
		delete[] LT[i];
	}


	delete[] LT;
	delete[] M;
	delete[] a;
	delete[] v;
	delete[] w;


	codi::RealForward result = wSvd * svd_multiplier *reg_term_svd + w12 * reg_term_L1;
	*regTerm = result.getValue();

	*regTermd = result.getGradient();

	return 0;


}



int calcRegTerms(float *L, float *Lb,float *result , float wSvd, float w12, int dim) {
	int flag, i, its, j, jj, k, l = 0, nm;


	codi::RealReverse *Lcodi = new codi::RealReverse[dim*dim];
	for (int i = 0; i < dim*dim; i++) {

		Lcodi[i] = L[i];

	}

	/* activate tape and register input */

	codi::RealReverse::TapeType& tape = codi::RealReverse::getGlobalTape();
	tape.setActive();

	codi::RealReverse regTerm=0.0;



	for (int i = 0; i < dim*dim; i++) {

		tape.registerInput(Lcodi[i]);

	}


	codi::RealReverse c, f, h, s, x, y, z;
	codi::RealReverse anorm = 0.0, g = 0.0, scale = 0.0;


	int m = dim;
	int n = dim;


	codi::RealReverse **a;
	a = new codi::RealReverse*[dim];

	for (i = 0; i < dim; i++) {

		a[i] = new codi::RealReverse[dim];
	}

	codi::RealReverse **M;
	M = new codi::RealReverse*[dim];

	for (i = 0; i < dim; i++) {

		M[i] = new codi::RealReverse[dim];
	}




	codi::RealReverse **LT;
	LT = new codi::RealReverse*[dim];
	for (int i = 0; i < dim; i++) {
		LT[i] = new codi::RealReverse[dim];

	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < dim; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = Lcodi[i*dim+j];
		}


	}
#if 0
	printf("LT = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", LT[i][j].getValue());

		}
		printf("\n");
	}

#endif

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=0;
			M[i][j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
			for(int k = 0; k < dim; ++k)
			{
				M[i][j] += Lcodi[i*dim+k] * LT[k][j];

			}

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{
			a[i][j]=M[i][j];

		}

#if 0
	printf("M = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", M[i][j].getValue());

		}
		printf("\n");
	}

#endif

#if 0
	printf("a = \n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", a[i][j].getValue());

		}
		printf("\n");
	}

#endif


	/* SVD part */

	codi::RealReverse **v;
	v = new codi::RealReverse*[n];

	for (i = 0; i < n; i++) {

		v[i] = new codi::RealReverse[n];
	}
	codi::RealReverse *w = new codi::RealReverse[n];

	codi::RealReverse *rv1 = new codi::RealReverse[n];

	/* Householder reduction to bidiagonal form */
	for (i = 0; i < n; i++) {
		/* left-hand reduction */
		l = i + 1;
		rv1[i] = scale * g;
		g = s = scale = 0.0;
		if (i < m) {
			for (k = i; k < m; k++)
				scale += fabs(a[k][i]);
			if (scale != 0) {
				for (k = i; k < m; k++) {
					a[k][i] = (a[k][i] / scale);
					s += (a[k][i] * a[k][i]);
				}
				f = a[i][i];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][i] = (f - g);
				if (i != n - 1) {
					for (j = l; j < n; j++) {
						for (s = 0.0, k = i; k < m; k++)
							s += (a[k][i] * a[k][j]);
						f = s / h;
						for (k = i; k < m; k++)
							a[k][j] += (f * a[k][i]);
					}
				}
				for (k = i; k < m; k++)
					a[k][i] = (a[k][i] * scale);
			}
		}
		w[i] = (scale * g);

		/* right-hand reduction */
		g = s = scale = 0.0;
		if (i < m && i != n - 1) {
			for (k = l; k < n; k++)
				scale += fabs(a[i][k]);
			if (scale !=0) {
				for (k = l; k < n; k++) {
					a[i][k] = (a[i][k] / scale);
					s += (a[i][k] * a[i][k]);
				}
				f = a[i][l];
				g = -SIGN(sqrt(s), f);
				h = f * g - s;
				a[i][l] = (f - g);
				for (k = l; k < n; k++)
					rv1[k] = a[i][k] / h;
				if (i != m - 1) {
					for (j = l; j < m; j++) {
						for (s = 0.0, k = l; k < n; k++)
							s += (a[j][k] * a[i][k]);
						for (k = l; k < n; k++)
							a[j][k] += (s * rv1[k]);
					}
				}
				for (k = l; k < n; k++)
					a[i][k] = (a[i][k] * scale);
			}
		}
		anorm = MAX(anorm, (fabs(w[i]) + fabs(rv1[i])));
	}

	/* accumulate the right-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		if (i < n - 1) {
			if (g !=0) {
				for (j = l; j < n; j++)
					v[j][i] = ((a[i][j] / a[i][l]) / g);
				/* float division to avoid underflow */
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < n; k++)
						s += (a[i][k] * v[k][j]);
					for (k = l; k < n; k++)
						v[k][j] += (s * v[k][i]);
				}
			}
			for (j = l; j < n; j++)
				v[i][j] = v[j][i] = 0.0;
		}
		v[i][i] = 1.0;
		g = rv1[i];
		l = i;
	}

	/* accumulate the left-hand transformation */
	for (i = n - 1; i >= 0; i--) {
		l = i + 1;
		g = w[i];
		if (i < n - 1)
			for (j = l; j < n; j++)
				a[i][j] = 0.0;
		if (g != 0) {
			g = 1.0 / g;
			if (i != n - 1) {
				for (j = l; j < n; j++) {
					for (s = 0.0, k = l; k < m; k++)
						s += (a[k][i] * a[k][j]);
					f = (s / a[i][i]) * g;
					for (k = i; k < m; k++)
						a[k][j] += (f * a[k][i]);
				}
			}
			for (j = i; j < m; j++)
				a[j][i] = (a[j][i] * g);
		} else {
			for (j = i; j < m; j++)
				a[j][i] = 0.0;
		}
		++a[i][i];
	}

	/* diagonalize the bidiagonal form */
	for (k = n - 1; k >= 0; k--) { /* loop over singular values */
		for (its = 0; its < 30000; its++) { /* loop over allowed iterations */
			flag = 1;
			for (l = k; l >= 0; l--) { /* test for splitting */
				nm = l - 1;
				if (fabs(rv1[l]) + anorm == anorm) {
					flag = 0;
					break;
				}
				if (fabs(w[nm]) + anorm == anorm)
					break;
			}
			if (flag) {
				c = 0.0;
				s = 1.0;
				for (i = l; i <= k; i++) {
					f = s * rv1[i];
					if (fabs(f) + anorm != anorm) {
						g = w[i];
						h = PYTHAG(f, g);
						w[i] = h;
						h = 1.0 / h;
						c = g * h;
						s = (-f * h);
						for (j = 0; j < m; j++) {
							y = a[j][nm];
							z = a[j][i];
							a[j][nm] = (y * c + z * s);
							a[j][i] = (z * c - y * s);
						}
					}
				}
			}
			z = w[k];
			if (l == k) { /* convergence */
				if (z < 0.0) { /* make singular value nonnegative */
					w[k] = (-z);
					for (j = 0; j < n; j++)
						v[j][k] = (-v[j][k]);
				}
				break;
			}
			if (its >= 30000) {
				delete[] rv1;
				fprintf(stderr, "No convergence after 30,000! iterations \n");
				return 1;
			}

			/* shift from bottom 2 x 2 minor */
			x = w[l];
			nm = k - 1;
			y = w[nm];
			g = rv1[nm];
			h = rv1[k];
			f = ((y - z) * (y + z) + (g - h) * (g + h)) / (2.0 * h * y);
			g = PYTHAG(f, 1.0);
			f = ((x - z) * (x + z) + h * ((y / (f + SIGN(g, f))) - h)) / x;

			/* next QR transformation */
			c = s = 1.0;
			for (j = l; j <= nm; j++) {
				i = j + 1;
				g = rv1[i];
				y = w[i];
				h = s * g;
				g = c * g;
				z = PYTHAG(f, h);
				rv1[j] = z;
				c = f / z;
				s = h / z;
				f = x * c + g * s;
				g = g * c - x * s;
				h = y * s;
				y = y * c;
				for (jj = 0; jj < n; jj++) {
					x = v[jj][j];
					z = v[jj][i];
					v[jj][j] = (x * c + z * s);
					v[jj][i] = (z * c - x * s);
				}
				z = PYTHAG(f, h);
				w[j] = z;
				if (z != 0) {
					z = 1.0 / z;
					c = f * z;
					s = h * z;
				}
				f = (c * g) + (s * y);
				x = (c * y) - (s * g);
				for (jj = 0; jj < m; jj++) {
					y = a[jj][j];
					z = a[jj][i];
					a[jj][j] = (y * c + z * s);
					a[jj][i] = (z * c - y * s);
				}
			}
			rv1[l] = 0.0;
			rv1[k] = f;
			w[k] = x;
		}
	}
	delete[] rv1;

#if 0
	printf("singular values of a=\n");

	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i]);
	}
#endif

	codi::RealReverse temp;
	for (i = 0; i < n; ++i) {
		for (j = i + 1; j < n; ++j) {

			if (w[i] < w[j])

			{
				temp = w[i];
				w[i] = w[j];
				w[j] = temp;
			}
		}
	}

#if 0
	printf("singular values of a=\n");


	for (i = 0; i < n; i++) {

		printf("%10.7f\n",w[i].getValue());
	}
#endif

	codi::RealReverse wsum = 0.0;
	for (i = 0; i < n; i++) {

		wsum += w[i];

	}

	for (i = 0; i < n; i++) {

		w[i] = w[i]/wsum;

	}

#if 0
	printf("singular values of a (normalized) with wsum =%10.7f\n",wsum.getValue());


	for (i = 0; i < n; i++) {

		printf("%15.10f\n",w[i].getValue());
	}
#endif


	codi::RealReverse svd_multiplier = (1.0*n*(1.0*n+1))/2.0;

	svd_multiplier = 1.0/svd_multiplier;
#if 0
	printf("svd_multiplier = %10.7f\n",svd_multiplier);
#endif
	codi::RealReverse reg_term_svd = 0.0;

	for (i = 0; i < n; i++) {
#if 0
		printf("%d * %10.7f = %10.7f\n",i+1,w[i].getValue(),(i+1)*w[i].getValue());
#endif
		reg_term_svd = reg_term_svd + (i + 1) * w[i];
	}
#if 0
	printf("reg_term_svd = %10.7f\n",reg_term_svd.getValue());
#endif


	codi::RealReverse reg_term_L1 = 0.0;

	for (i = 0; i < n; i++)
		for (j = 0; j < n; j++) {

			reg_term_L1 = reg_term_L1 + M[i][j]* M[i][j];
		}
#if 0
	printf("reg_term_L1 = %10.7f\n",reg_term_L1.getValue());
#endif







	regTerm = wSvd * svd_multiplier *reg_term_svd + w12 * reg_term_L1;

#if 0
	printf("w12 * reg_term_L1 = %10.7f\n",w12 * reg_term_L1.getValue());
#endif


	tape.registerOutput(regTerm);

	tape.setPassive();
	regTerm.setGradient(1.0);
	tape.evaluate();

	for (int i = 0; i < numVar*numVar; i++) {


		Lb[i] = Lcodi[i].getGradient();

	}


	tape.reset();

	*result = regTerm.getValue(); 

	for (i = 0; i < n; i++) {

		delete[] v[i];
		delete[] a[i];
		delete[] M[i];
		delete[] LT[i];
	}


	delete[] LT;
	delete[] M;
	delete[] a;
	delete[] v;
	delete[] w;
	delete[] Lcodi;


	return 0;




}


/** calculate regularization terms for the given matrix L
 *
 * @param[in]  L:  lower diagonal matrix
 * @param[in]  w12: weight for the mixed 12 regularization part
 * @param[out] regTerm
 *
 */

int calcRegTermL12(float *L, float *regTerm, float w12, int dim) {




	float **M;
	M= new float*[dim];

	for (int i = 0; i < dim; i++) {

		M[i] = new float[dim];
	}


	float **LT;
	LT = new float*[dim];
	for (int i = 0; i < dim; i++) {
		LT[i] = new float[dim];

	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < dim; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = L[i*dim+j];
		}


	}



	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{

			M[i][j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
			for(int k = 0; k < dim; ++k)
			{
				M[i][j] += L[i*dim+k] * LT[k][j];

			}


	float reg_term_L1 = 0.0;

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			reg_term_L1 = reg_term_L1 + M[i][j]* M[i][j];
		}
#if 0
	printf("reg_term_L1 = %10.7f\n",reg_term_L1);
#endif



	for (int i = 0; i < dim; i++) {
		delete[] M[i];
		delete[] LT[i];
	}


	delete[] LT;
	delete[] M;


	*regTerm = w12 * reg_term_L1;
#if 0
	printf("result = %10.7f\n",*regTerm);
#endif

	return 0;




}


int calcRegTermL12(float *L, float *Lb,float *result , float w12, int dim) {



	codi::RealReverse *Lcodi = new codi::RealReverse[dim*dim];
	for (int i = 0; i < dim*dim; i++) {

		Lcodi[i] = L[i];

	}

	/* activate tape and register input */

	codi::RealReverse::TapeType& tape = codi::RealReverse::getGlobalTape();
	tape.setActive();

	codi::RealReverse regTerm=0.0;



	for (int i = 0; i < dim*dim; i++) {

		tape.registerInput(Lcodi[i]);

	}





	codi::RealReverse **M;
	M = new codi::RealReverse*[dim];

	for (int i = 0; i < dim; i++) {

		M[i] = new codi::RealReverse[dim];
	}




	codi::RealReverse **LT;
	LT = new codi::RealReverse*[dim];
	for (int i = 0; i < dim; i++) {
		LT[i] = new codi::RealReverse[dim];

	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < dim; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = Lcodi[i*dim+j];
		}


	}

	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
		{

			M[i][j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < dim; ++i)
		for(int j = 0; j < dim; ++j)
			for(int k = 0; k < dim; ++k)
			{
				M[i][j] += Lcodi[i*dim+k] * LT[k][j];

			}







	codi::RealReverse reg_term_L1 = 0.0;

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			reg_term_L1 = reg_term_L1 + M[i][j]* M[i][j];
		}


	regTerm = w12 * reg_term_L1;

#if 0
	printf("w12 * reg_term_L1 = %10.7f\n",w12 * reg_term_L1.getValue());
#endif


	tape.registerOutput(regTerm);

	tape.setPassive();
	regTerm.setGradient(1.0);
	tape.evaluate();

	for (int i = 0; i < numVar*numVar; i++) {


		Lb[i] = Lcodi[i].getGradient();

	}


	tape.reset();

	*result = regTerm.getValue();

	for (int i = 0; i < dim; i++) {

		delete[] M[i];
		delete[] LT[i];
	}


	delete[] LT;
	delete[] M;
	delete[] Lcodi;


	return 0;




}





float calcKernelValCPU(rowvec &xi, rowvec &xj, mat &M, float sigma){

	rowvec diff = xi - xj;
	colvec diffT = trans(diff);

	vec matVecProd = M * diffT;
	//	printf("M * xdiff = \n");
	//	matVecProd.print();

	float metricVal = dot(diff, M * diffT);

	float sqr_two_pi = sqrt(2.0 * 3.14159265359);

	float kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));


	return (kernelVal);



}
/*
 * calculates the generalized Mahalanobis distance between two points
 *
 * @param[in] x_i : first vector
 * @param[in] X_j : second vector
 * @param[in] M : dim x dim matrix
 * @param[in] dim
 * @return distance
 *
 * */

float calcMetric(float *xi, float *xj, float *M, int dim) {

#if 0
	printf("calling calcMetric (primal)...\n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", M[i*dim+j]);

		}
		printf("\n");
	}

#endif



	float *diff = new float[dim];

	for (int i = 0; i < dim; i++) {

		diff[i] = xi[i] - xj[i];
	}
#if 0
	rowvec xi_val(dim);
	rowvec xj_val(dim);
	rowvec diff_val(dim);
	mat M_val(dim, dim);

	for (int i = 0; i < dim; i++) {
		xi_val(i) = xi[i];
		xj_val(i) = xj[i];
	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++)
			M_val(i, j) = M[i][j];

	diff_val = xi_val - xj_val;

	printf("diff_val=\n");
	diff_val.print();

	colvec diffT = trans(diff_val);

	vec matVecProd = M_val * diffT;
	printf("M * xdiff = \n");
	matVecProd.print();

	float metric_val = dot(diff_val, M_val * diffT);

	printf("metric_val = %10.7f\n", metric_val);
#endif

	float *tempVec = new float[dim];

	float sum = 0.0;

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			sum = sum + M[i*dim+j] * diff[j];
		}

		tempVec[i] = sum;
		sum = 0.0;

	}
#if 0
	printf("tempVec = \n");
	for(int i=0; i<dim; i++) {
		printf("%10.7f \n",tempVec[i] );

	}
#endif

	sum = 0.0;

	for (int i = 0; i < dim; i++) {

		sum = sum + tempVec[i] * diff[i];
	}
#if 0
	printf("sum = %10.7f\n",sum);
#endif

	delete[] diff;
	delete[] tempVec;


	if (sum < 0.0) {

		fprintf(stderr, "Error: metric is negative! at FILE = %s, LINE = %d.\n",__FILE__, __LINE__);
		exit(-1);
	}

	return sum;

}

/*
 * calculates the generalized Mahalanobis distance between two points, codiPack reverse mode
 * (differentiated in reverse mode )
 * @param[in] x_i : first vector
 * @param[in] X_j : second vector
 * @param[in] M : dim x dim matrix
 * @param[in] dim
 * @return distance
 *
 * */

codi::RealReverse calcMetric(float *xi, float *xj, codi::RealReverse *M,
		int dim) {

#if 0
	printf("calling calcMetric (adjoint)...\n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", M[i*dim+j].getValue());

		}
		printf("\n");
	}

#endif

	codi::RealReverse *diff = new codi::RealReverse[dim];

	for (int i = 0; i < dim; i++) {

		diff[i] = xi[i] - xj[i];
	}
#if 0
	rowvec xi_val(dim);
	rowvec xj_val(dim);
	rowvec diff_val(dim);
	mat M_val(dim, dim);

	for (int i = 0; i < dim; i++) {
		xi_val(i) = xi[i];
		xj_val(i) = xj[i];
	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			M_val(i, j) = M[i*dim+j].getValue();
		}

	diff_val = xi_val - xj_val;

	printf("diff_val=\n");
	diff_val.print();

	colvec diffT = trans(diff_val);

	vec matVecProd = M_val * diffT;
	printf("M * xdiff = \n");
	matVecProd.print();

	float metric_val = dot(diff_val, M_val * diffT);

	printf("metric_val = %10.7f\n", metric_val);
#endif

	codi::RealReverse *tempVec = new codi::RealReverse[dim];

	codi::RealReverse sum = 0.0;

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			sum = sum + M[i*dim+j] * diff[j];
		}

		tempVec[i] = sum;
		sum = 0.0;

	}
#if 0
	printf("tempVec = \n");
	for (int i = 0; i < dim; i++) {
		printf("%10.7f \n", tempVec[i].getValue());

	}
#endif

	sum = 0.0;

	for (int i = 0; i < dim; i++) {

		sum = sum + tempVec[i] * diff[i];
	}
#if 0
	printf("sum = %10.7f\n", sum.getValue());
#endif

	delete[] diff;
	delete[] tempVec;


	if (sum < 0.0) {

		fprintf(stderr, "Error: metric is negative! at %s, line %d.\n",__FILE__, __LINE__);
		fprintf(stderr, "metric val = %10.7f\n",sum.getValue());


		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}


		exit(-1);
	}

	return sum;

}

codi::RealForward calcMetric(float *xi, float *xj, codi::RealForward *M,
		int dim) {

#if 0
	printf("calling calcMetric (adjoint)...\n");

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			printf("%10.7f ", M[i*dim+j].getValue());

		}
		printf("\n");
	}

#endif

	codi::RealForward *diff = new codi::RealForward[dim];

	for (int i = 0; i < dim; i++) {

		diff[i] = xi[i] - xj[i];
	}
#if 0
	rowvec xi_val(dim);
	rowvec xj_val(dim);
	rowvec diff_val(dim);
	mat M_val(dim, dim);

	for (int i = 0; i < dim; i++) {
		xi_val(i) = xi[i];
		xj_val(i) = xj[i];
	}

	for (int i = 0; i < dim; i++)
		for (int j = 0; j < dim; j++) {

			M_val(i, j) = M[i*dim+j].getValue();
		}

	diff_val = xi_val - xj_val;

	printf("diff_val=\n");
	diff_val.print();

	colvec diffT = trans(diff_val);

	vec matVecProd = M_val * diffT;
	printf("M * xdiff = \n");
	matVecProd.print();

	float metric_val = dot(diff_val, M_val * diffT);

	printf("metric_val = %10.7f\n", metric_val);
#endif

	codi::RealForward *tempVec = new codi::RealForward[dim];

	codi::RealForward sum = 0.0;

	for (int i = 0; i < dim; i++) {
		for (int j = 0; j < dim; j++) {

			sum = sum + M[i*dim+j] * diff[j];
		}

		tempVec[i] = sum;
		sum = 0.0;

	}
#if 0
	printf("tempVec = \n");
	for (int i = 0; i < dim; i++) {
		printf("%10.7f \n", tempVec[i].getValue());

	}
#endif

	sum = 0.0;

	for (int i = 0; i < dim; i++) {

		sum = sum + tempVec[i] * diff[i];
	}
#if 0
	printf("sum = %10.7f\n", sum.getValue());
#endif

	delete[] diff;
	delete[] tempVec;


	if (sum < 0.0) {

		fprintf(stderr, "Error: metric is negative! at %s, line %d.\n",__FILE__, __LINE__);
		fprintf(stderr, "metric val = %10.7f\n",sum.getValue());


		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}


		exit(-1);
	}

	return sum;

}


float gaussianKernel(float *xi, float *xj, float sigma, float *M,
		int dim) {

#if 0
	printf("calling gaussianKernel...\n");
#endif

	/* calculate distance between xi and xj with the matrix M */
	float metricVal = calcMetric(xi, xj, M, dim);
#if 0
	printf("metricVal = %10.7f\n",metricVal);
#endif

	float sqr_two_pi = sqrt(2.0 * datum::pi);

	float kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));
#if 0
	printf("kernelVal = %10.7f\n",kernelVal);

#endif

	if(isnan(kernelVal)){

		fprintf(stderr, "Error: kernel value is NaN! at %s, line %d.\n",__FILE__, __LINE__);
		exit(-1);
	}

	if(kernelVal < 0.0){

		fprintf(stderr, "Error: kernel value is negative! at %s, line %d.\n",__FILE__, __LINE__);
		exit(-1);
	}

	kernelVal += 10E-14;


	return kernelVal;

}

codi::RealReverse gaussianKernel(float *xi, float *xj,
		codi::RealReverse sigma, codi::RealReverse *M, int dim) {

#if 0
	printf("calling gaussianKernel...\n");
#endif

	/* calculate distance between xi and xj with the matrix M */
	codi::RealReverse metricVal = calcMetric(xi, xj, M, dim);
#if 0
	printf("metricVal = %10.7f\n",metricVal.getValue());
#endif

	float sqr_two_pi = sqrt(2.0 * datum::pi);

	codi::RealReverse kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));

	if(isnan(kernelVal.getValue())){

		fprintf(stderr, "Error: kernel value is NaN! at %s, line %d.\n",__FILE__, __LINE__);

		printf("sigma = %10.7f\n",sigma.getValue());

		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}


		exit(-1);
	}



	kernelVal += 10E-14;
	//	printf("EPSILON = %10.7f ", EPSILON);

	if(kernelVal.getValue() < 0.0){

		fprintf(stderr, "Error: kernel value is negative or zero! at %s, line %d.\n",__FILE__, __LINE__);
		fprintf(stderr, "kernelVal = %20.15f\n",kernelVal.getValue() );
		fprintf(stderr, "metric val = %20.15f\n",metricVal.getValue());
		fprintf(stderr, "sigma = %20.15f\n",sigma.getValue());
		fprintf(stderr, "exp(-metricVal / (2 * sigma * sigma)) = %20.15f\n",exp(-metricVal / (2 * sigma * sigma)).getValue());



		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}



		exit(-1);
	}


#if 0
	printf("kernelVal = %10.7f\n",kernelVal.getValue());
#endif
	return kernelVal;

}


codi::RealForward gaussianKernel(float *xi, float *xj,
		codi::RealForward sigma, codi::RealForward *M, int dim) {


	/* calculate distance between xi and xj with the matrix M */
	codi::RealForward metricVal = calcMetric(xi, xj, M, dim);

	float sqr_two_pi = sqrt(2.0 * datum::pi);

	codi::RealForward kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-metricVal / (2 * sigma * sigma));

	if(isnan(kernelVal.getValue())){

		fprintf(stderr, "Error: kernel value is NaN! at %s, line %d.\n",__FILE__, __LINE__);

		printf("sigma = %10.7f\n",sigma.getValue());

		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}


		exit(-1);
	}



	kernelVal += 10E-14;
	//	printf("EPSILON = %10.7f ", EPSILON);

	if(kernelVal.getValue() < 0.0){

		fprintf(stderr, "Error: kernel value is negative or zero! at %s, line %d.\n",__FILE__, __LINE__);
		fprintf(stderr, "kernelVal = %20.15f\n",kernelVal.getValue() );
		fprintf(stderr, "metric val = %20.15f\n",metricVal.getValue());
		fprintf(stderr, "sigma = %20.15f\n",sigma.getValue());
		fprintf(stderr, "exp(-metricVal / (2 * sigma * sigma)) = %20.15f\n",exp(-metricVal / (2 * sigma * sigma)).getValue());



		printf("M = \n");

		for (int i = 0; i < dim; i++) {
			for (int j = 0; j < dim; j++) {

				printf("%10.7f ", M[i*dim+j].getValue());

			}
			printf("\n");
		}



		exit(-1);
	}


#if 0
	printf("kernelVal = %10.7f\n",kernelVal.getValue());
#endif
	return kernelVal;

}

void calcLossFunCPU(float *result, float *input, float *data,int N){

	float LT[numVar][numVar];
	float L[numVar][numVar];
	float M[numVar*numVar+1];

	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			L[i][j] = input[i*numVar + j];




	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			LT[i][j] = 0.0;


	for (int i = 0; i < numVar; ++i) {
		for (int j = 0; j < i+1; ++j)
			LT[j][i] = L[i][j];
	}



	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			M[i*numVar + j] = 0;



	/* Multiplying matrix L and LT and storing in M */
	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			for (int k = 0; k < numVar; ++k)
				M[i*numVar + j] = M[i*numVar + j] + L[i][k]*LT[k][j];



	M[numVar*numVar] = input[numVar*numVar];

	float sigma = M[numVar*numVar];

	float *xp = new float[numVar];
	float *xi = new float[numVar];

	float *kernelVal = new float[N];

	float lossFunc = 0.0;

	for (int i = 0; i < N; i++) {

#if 0
		printf("kernel regression for the sample number %d\n",i);

#endif
		for (int k = 0; k < numVar; k++) {

			xp[k] = data[i*(numVar+1)+k];
		}

		float kernelSum = 0.0;
		for (int j = 0; j < N; j++) {

			if (i != j) {

				for (int k = 0; k < numVar; k++) {

					xi[k] = data[j*(numVar+1)+k];
				}
				kernelVal[j] = gaussianKernel(xi, xp, sigma, M, numVar);
				kernelSum += kernelVal[j];
#if 0
				printf("kernelVal[%d]=%10.7f\n",j,kernelVal[j]);
#endif
			}
		}

		float fApprox = 0.0;
		for (int j = 0; j < N; j++) {
			if (i != j) {
				fApprox += kernelVal[j] * data[j*(numVar+1)+numVar];

			}
		}

		fApprox = fApprox / kernelSum;

#if 0
		printf("fApprox = %10.7f\n",fApprox);
		printf("fExact = %10.7f\n",data[i*(numVar+1)+numVar]);
#endif

		lossFunc += (fApprox - data[i*(numVar+1)+numVar]) * (fApprox - data[i*(numVar+1)+numVar]);

	} // end of i loop

	lossFunc = lossFunc / N;

	*result = lossFunc; 

	delete[] xp;
	delete[] xi;
	delete[] kernelVal;


}



void calcLossFunCPU(codi::RealReverse *result, codi::RealReverse *input, float *inputb,float *data,int N){


	/* activate tape and register input */

	codi::RealReverse::TapeType& tape = codi::RealReverse::getGlobalTape();
	tape.setActive();



	for (int i = 0; i < numVar*numVar+1; i++) {
		tape.registerInput(input[i]);

	}




	codi::RealReverse LT[numVar][numVar];
	codi::RealReverse L[numVar][numVar];
	codi::RealReverse M[numVar*numVar+1];

	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			M[i*numVar + j] = 0;





	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			L[i][j] = input[i*numVar + j];

#if 0
	printf("L = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",L[i][j].getValue());

		}
		printf("\n");
	}
#endif	


	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j) {

			LT[i][j] = 0.0;
		}





	for (int i = 0; i < numVar; ++i) {
		for (int j = 0; j < i+1; ++j){


			LT[j][i] = L[i][j];
		}
	}

#if 0
	printf("LT = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",LT[i][j].getValue());

		}
		printf("\n");
	}
#endif	









	/* Multiplying matrix L and LT and storing in M */
	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			for (int k = 0; k < numVar; ++k) {

				M[i*numVar + j] = M[i*numVar + j] + L[i][k]*LT[k][j];
			}



#if 0
	printf("M = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",M[i*numVar + j].getValue());

		}
		printf("\n");
	}
#endif	



	M[numVar*numVar] = input[numVar*numVar];



	codi::RealReverse sigma = M[numVar*numVar]; 

	float *xi = new float[numVar];
	float *xj = new float[numVar];

	codi::RealReverse **kernelValTable = new codi::RealReverse*[N];

	for(int i=0; i<N;i++) {

		kernelValTable[i] = new codi::RealReverse[N];

	}

	for (int i = 0; i < N; i++) 

		for (int j = 0; j < N; j++) kernelValTable[i][j] = 0.0;



	for (int i = 0; i < N; i++) {

		for (int j = i+1; j < N; j++) {



			for (int k = 0; k < numVar; k++) {

				xi[k] = data[i*(numVar+1)+k];
				xj[k] = data[j*(numVar+1)+k];
			}


			kernelValTable[i][j] = gaussianKernel(xi, xj, sigma, M, numVar);
			kernelValTable[j][i] = kernelValTable[i][j]; 
			//			printf("%d kernelValTable[%d][%d] = %10.7f\n",i*N+j,i,j,kernelValTable[i][j].getValue());





		}



	}

	codi::RealReverse lossFunc = 0.0;

	for (int i = 0; i < N; i++) {

#if 0
		printf("kernel regression for the sample number %d\n",i);

#endif


		codi::RealReverse kernelSum = 0.0;
		for (int j = 0; j < N; j++) {

			if (i != j) {


				kernelSum += kernelValTable[i][j];

			}
		}

		codi::RealReverse fApprox = 0.0;
		for (int j = 0; j < N; j++) {
			if (i != j) {
				fApprox += kernelValTable[i][j] * data[j*(numVar+1)+numVar];

			}
		}

		fApprox = fApprox / kernelSum;

#if 0
		printf("fApprox = %10.7f\n",fApprox.getValue());
		printf("fExact = %10.7f\n",data[i*(numVar+1)+numVar]);
#endif

		lossFunc += (fApprox - data[i*(numVar+1)+numVar]) * (fApprox - data[i*(numVar+1)+numVar]);

	} // end of i loop

	lossFunc = lossFunc / N;


#if 1

	printf("lossFunc (reverse mode CodiPack) = %10.7f\n",lossFunc.getValue());
#endif	

	*result = lossFunc; 


	tape.registerOutput(*result);

	tape.setPassive();
	result->setGradient(1.0);
	tape.evaluate();

#if 0
	printf("Mb = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",M[i*numVar + j].getGradient());

		}
		printf("\n");
	}
#endif	




	for (int i = 0; i < numVar*numVar+1; i++) {


		inputb[i] = input[i].getGradient();

	}


	tape.reset();

	delete[] xi;
	delete[] xj;

	for(int i=0; i<N;i++) {

		delete[] kernelValTable[i];

	}

	delete[] kernelValTable;




}

void calcLossFunCPU(codi::RealForward *result, codi::RealForward *input,int tldIndx, float *data,int N){

	input[tldIndx].setGradient(1.0);



	codi::RealForward LT[numVar][numVar];
	codi::RealForward L[numVar][numVar];
	codi::RealForward M[numVar*numVar+1];

	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			L[i][j] = input[i*numVar + j];

#if 0
	printf("L = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",L[i][j].getValue());

		}
		printf("\n");
	}
#endif	


	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j) {

			LT[i][j] = 0.0;
		}





	for (int i = 0; i < numVar; ++i) {
		for (int j = 0; j < i+1; ++j){


			LT[j][i] = L[i][j];
		}
	}

#if 0
	printf("LT = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",LT[i][j].getValue());

		}
		printf("\n");
	}
#endif	


	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			M[i*numVar + j] = 0;






	/* Multiplying matrix L and LT and storing in M */
	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			for (int k = 0; k < numVar; ++k) {

				M[i*numVar + j] = M[i*numVar + j] + L[i][k]*LT[k][j];
			}


#if 0
	printf("M = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",M[i*numVar + j].getValue());

		}
		printf("\n");
	}
#endif	



	M[numVar*numVar] = input[numVar*numVar];

	codi::RealForward sigma = M[numVar*numVar]; 

	float *xi = new float[numVar];
	float *xj = new float[numVar];

	codi::RealForward **kernelValTable = new codi::RealForward*[N];

	for(int i=0; i<N;i++) {

		kernelValTable[i] = new codi::RealForward[N];

	}

	for (int i = 0; i < N; i++) 

		for (int j = 0; j < N; j++) kernelValTable[i][j] = 0.0;




	for (int i = 0; i < N; i++) {

		for (int j = 0; j < N; j++) {

			if(j>i){

				for (int k = 0; k < numVar; k++) {

					xi[k] = data[i*(numVar+1)+k];
					xj[k] = data[j*(numVar+1)+k];
				}


				kernelValTable[i][j] = gaussianKernel(xi, xj, sigma, M, numVar);

			}



		}



	}



	codi::RealForward lossFunc = 0.0;

	for (int i = 0; i < N; i++) {

#if 0
		printf("kernel regression for the sample number %d\n",i);

#endif


		codi::RealForward kernelSum = 0.0;
		for (int j = 0; j < N; j++) {

			if (i != j) {


				kernelSum += kernelValTable[i][j];

			}
		}

		codi::RealForward fApprox = 0.0;
		for (int j = 0; j < N; j++) {
			if (i != j) {
				fApprox += kernelValTable[i][j] * data[j*(numVar+1)+numVar];

			}
		}

		fApprox = fApprox / kernelSum;

#if 0
		printf("fApprox = %10.7f\n",fApprox.getValue());
		printf("fExact = %10.7f\n",data[i*(numVar+1)+numVar]);
#endif

		lossFunc += (fApprox - data[i*(numVar+1)+numVar]) * (fApprox - data[i*(numVar+1)+numVar]);

	} // end of i loop

	lossFunc = lossFunc / N;


#if 0

	printf("lossFunc = %10.7f\n",lossFunc.getValue());
#endif	

	*result = lossFunc; 
	delete[] xi;
	delete[] xj;

	for(int i=0; i<N;i++) {

		delete[] kernelValTable[i];

	}

	delete[] kernelValTable;


}




__global__ void calculateKernelValues_b(float *ab, float *X, float *kernelValTable, float *kernelValTableb, int N) {


	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	float sigma = MDevice[numVar*numVar];
	float sigmab = 0.0;
	/* calculate column index */
	int indx2 = tid%N;
	/* calculate row index */
	int indx1 = tid/N;



	if (indx2 > indx1) {
		int off1 = indx1*(numVar+1);
		int off2 = indx2*(numVar+1);

		float diff[numVar];

		float tempVec[numVar];
		float tempVecb[numVar];

		float sumb = 0.0;


		float kernelValb = 0.0;
		float temp;
		float temp0;
		float tempb;
		float tempb0;
		for (int k = 0; k < numVar; ++k)
			diff[k] = X[off1 + k] - X[off2 + k];

		float sum = 0.0;
		for (int i = 0; i < numVar; ++i) {
			for (int j = 0; j < numVar; ++j)
				sum = sum + MDevice[i*numVar+j]*diff[j];
			tempVec[i] = sum;
			sum = 0.0;
		}
		sum = 0.0;
		for (int i = 0; i < numVar; ++i)
			sum = sum + tempVec[i]*diff[i];

		float sqr_two_pi;
		sqr_two_pi = sqrt(2.0*3.14159265359);
		float kernelVal = 1.0/(sigma*sqr_two_pi)*exp(-sum/(2*sigma*sigma))+10E-12;


		kernelValb = kernelValTableb[indx1*N + indx2];


		kernelValTableb[indx1*N + indx2] = 0.0;
		tempb = kernelValb/(sqr_two_pi*sigma);





		temp = 2*(sigma*sigma);
		temp0 = sum/temp; // temp0 = sum/2*(sigma*sigma)
		tempb0 = -(exp(-temp0)*tempb/temp); // -(exp(-sum/2*(sigma*sigma))*kernelValb/(sqr_two_pi*sigma)/temp) 
		sumb = tempb0;
		sigmab = -(exp(-temp0)*tempb/sigma) - 2*2*temp0*sigma*tempb0;




		for (int i = 0; i < numVar; ++i){

			tempVecb[i] = 0.0;

		}

		for (int i = numVar-1; i > -1; --i){

			tempVecb[i] = tempVecb[i] + diff[i]*sumb;
		}

		for (int i = numVar-1; i > -1; --i) {
			sumb = tempVecb[i];
			tempVecb[i] = 0.0;
			for (int j = numVar-1; j > -1; --j){

				float addTerm = diff[j]*sumb;


				atomicAdd( &ab[i*numVar + j],addTerm );

			}
			//				ab[i*numVar + j] = ab[i*numVar + j] + diff[j]*sumb;
		}
	} 
	atomicAdd( &ab[numVar*numVar],sigmab );
	//	ab[numVar*numVar] = ab[numVar*numVar] + sigmab;


}



__global__ void calculateKernelValues(float *X, float *kernelValTable, int N){


	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	float sigma = MDevice[numVar*numVar];

	/* calculate column index */
	int indx2 = tid%N;

	/* calculate row index */
	int indx1 = tid/N;

	if(indx2 > indx1){

		int off1 = indx1*(numVar+1);
		int off2 = indx2*(numVar+1);

		float diff[numVar];

		for (int k = 0; k < numVar; k++) {

			diff[k] = X[off1+k] - X[off2+k];

		}


		float tempVec[numVar];
		float sum = 0.0;

		for (int i = 0; i < numVar; i++) {
			for (int j = 0; j < numVar; j++) {

				sum = sum + MDevice[i*numVar+j] * diff[j];
			}

			tempVec[i] = sum;
			sum = 0.0;

		}


		sum = 0.0;

		for (int i = 0; i < numVar; i++) {

			sum = sum + tempVec[i] * diff[i];
		}




		float sqr_two_pi = sqrt(2.0 * 3.14159265359);

		float kernelVal = (1.0 / (sigma * sqr_two_pi))* exp(-sum / (2 * sigma * sigma)) + 10E-12;



		kernelValTable[indx1*N+indx2]= kernelVal;

	}



}

__global__  void calculateLossKernelL1(float *X,float *kernelValTable, float *sum, int N){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < N){


		float lossFunc = 0.0;

		float kernelSum = 0.0;

		for(int i=0; i<N; i++){

			if(tid != i){

				int indxKernelValTable;
				if(i<tid) {


					indxKernelValTable = i*N+tid;

				}
				else{

					indxKernelValTable = tid*N+i;

				}

				kernelSum += kernelValTable[indxKernelValTable];

			}



		}

		float fapprox=0.0;
		for(int i=0; i<N; i++){

			if(tid != i){
				int indxKernelValTable;

				if(i<tid) {

					indxKernelValTable = i*N+tid;

				}
				else{

					indxKernelValTable = tid*N+i;

				}

				fapprox += (kernelValTable[indxKernelValTable]/kernelSum)* X[i*(numVar+1)+numVar];

			}




		}


		//		lossFunc = (fapprox - X[tid*(numVar+1)+numVar]) * (fapprox - X[tid*(numVar+1)+numVar]);
		lossFunc = fabs(fapprox - X[tid*(numVar+1)+numVar]);
		sum[tid] = lossFunc;
	}

}

__global__  void calculateLossKernelL2(float *X,float *kernelValTable, float *sum, int N){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < N){


		float lossFunc = 0.0;

		float kernelSum = 0.0;

		for(int i=0; i<N; i++){

			if(tid != i){

				int indxKernelValTable;
				if(i<tid) {


					indxKernelValTable = i*N+tid;

				}
				else{

					indxKernelValTable = tid*N+i;

				}

				kernelSum += kernelValTable[indxKernelValTable];

			}



		}

		float fapprox=0.0;
		for(int i=0; i<N; i++){

			if(tid != i){
				int indxKernelValTable;

				if(i<tid) {

					indxKernelValTable = i*N+tid;

				}
				else{

					indxKernelValTable = tid*N+i;

				}

				fapprox += (kernelValTable[indxKernelValTable]/kernelSum)* X[i*(numVar+1)+numVar];

			}




		}


		lossFunc = (fapprox - X[tid*(numVar+1)+numVar]) * (fapprox - X[tid*(numVar+1)+numVar]);

		sum[tid] = lossFunc;
	}

}



__global__  void calculateLossKernelL1_b(float *X, float *kernelValTable, float *
		kernelValTableb, float *sum, float *sumb, int N) {


	int tid = threadIdx.x + blockIdx.x * blockDim.x;


	if (tid < N) {


		float lossFunc;
		float lossFuncb;
		float kernelSum=0.0;
		float kernelSumb;

		float fapproxb;


		for (int i = 0; i < N; ++i){
			if (tid != i) {
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;
				kernelSum = kernelSum + kernelValTable[indxKernelValTable];


			}
		}


		float fapprox = 0.0;
		for (int i = 0; i < N; ++i){
			if (tid != i) {
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;
				fapprox = fapprox + kernelValTable[indxKernelValTable]/
						kernelSum*X[i*(numVar+1)+numVar];
#if 0
				if (isnan (fapprox ) || isinf (fapprox) ){

					printf("fapprox  is NaN or inf %10.7f\n",kernelSum);

					assert(0);

				}
#endif

			}
		}




		//		lossFunc = (fapprox - X[tid*(numVar+1)+numVar]) * (fapprox - X[tid*(numVar+1)+numVar]);
		lossFunc = fabs ( (fapprox - X[tid*(numVar+1)+numVar]) );


		sum[tid] = lossFunc;

		lossFuncb = sumb[tid];
#if 0
		if (isnan (sumb[tid] ) || isinf (sumb[tid]) ){

			printf("sumb[tid]  is NaN or inf!\n");



		}
#endif


		sumb[tid] = 0.0;
		//		fapproxb = 2*(fapprox-X[tid*(numVar+1)+numVar])*lossFuncb;

		if((fapprox - X[tid*(numVar+1)+numVar]) >= 0){


			fapproxb = lossFuncb;

		}
		else{

			fapproxb = -lossFuncb;

		}



		kernelSumb = 0.0;
		for (int i = N-1; i > -1; --i) {

			if (tid != i)  {
				float tempb;
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;



				tempb = X[i*(numVar+1)+numVar]*fapproxb/kernelSum;

				kernelValTableb[indxKernelValTable] = kernelValTableb[indxKernelValTable] + tempb;
				kernelSumb = kernelSumb - kernelValTable[indxKernelValTable]*
						tempb/kernelSum;
			}
		}
		for (int i = N-1; i > -1; --i) {

			if (tid != i)  {
				int indxKernelValTable;

				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;

				kernelValTableb[indxKernelValTable] = kernelValTableb[indxKernelValTable] + kernelSumb;



			}
		}


	}

}


__global__  void calculateLossKernelL2_b(float *X, float *kernelValTable, float *
		kernelValTableb, float *sum, float *sumb, int N) {


	int tid = threadIdx.x + blockIdx.x * blockDim.x;


	if (tid < N) {


		float lossFunc;
		float lossFuncb;
		float kernelSum=0.0;
		float kernelSumb;

		float fapproxb;


		for (int i = 0; i < N; ++i){
			if (tid != i) {
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;
				kernelSum = kernelSum + kernelValTable[indxKernelValTable];


			} 
		}


		float fapprox = 0.0;
		for (int i = 0; i < N; ++i){
			if (tid != i) {
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;
				fapprox = fapprox + kernelValTable[indxKernelValTable]/
						kernelSum*X[i*(numVar+1)+numVar];
#if 0
				if (isnan (fapprox ) || isinf (fapprox) ){

					printf("fapprox  is NaN or inf %10.7f\n",kernelSum);

					assert(0);

				}
#endif			

			} 
		}




		lossFunc = (fapprox - X[tid*(numVar+1)+numVar]) * (fapprox - X[tid*(numVar+1)+numVar]);


		sum[tid] = lossFunc;

		lossFuncb = sumb[tid];
#if 0
		if (isnan (sumb[tid] ) || isinf (sumb[tid]) ){

			printf("sumb[tid]  is NaN or inf!\n");



		}
#endif				


		sumb[tid] = 0.0;
		fapproxb = 2*(fapprox-X[tid*(numVar+1)+numVar])*lossFuncb;




		kernelSumb = 0.0;
		for (int i = N-1; i > -1; --i) {

			if (tid != i)  {
				float tempb;
				int indxKernelValTable;
				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;



				tempb = X[i*(numVar+1)+numVar]*fapproxb/kernelSum;

				kernelValTableb[indxKernelValTable] = kernelValTableb[indxKernelValTable] + tempb;
				kernelSumb = kernelSumb - kernelValTable[indxKernelValTable]*
						tempb/kernelSum;
			}
		}
		for (int i = N-1; i > -1; --i) {

			if (tid != i)  {
				int indxKernelValTable;

				if (i < tid)
					indxKernelValTable = i*N + tid;
				else
					indxKernelValTable = tid*N + i;

				kernelValTableb[indxKernelValTable] = kernelValTableb[indxKernelValTable] + kernelSumb;



			}
		}


	} 

}


void calcLossFunGPU(float *result, float *input, float *data,int N, int lossFunType){

	hipEvent_t start, stop;
	hipEventCreate( &start ) ;
	hipEventCreate( &stop ) ;
	hipEventRecord( start, 0 ) ;

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;


	float LT[numVar][numVar];
	float L[numVar][numVar];
	float M[numVar*numVar+1];



	for (int i = 0; i < numVar; i++)
		for (int j = 0; j < numVar; j++) {
			L[i][j]=input[i*numVar+j];

		}


#if 1
	printf("Data (host) = \n");

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < numVar+1; j++) {

			printf("%10.7f ", data[i*(numVar+1)+j]);

		}
		printf("\n");
	}

#endif	



#if 1
	printf("L = \n");

	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ", L[i][j]);

		}
		printf("\n");
	}

#endif




	for (int i = 0; i < numVar; i++)
		for (int j = 0; j < numVar; j++) {

			LT[i][j]=0.0;
		}




	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j <= i; j++){

			LT[j][i] = L[i][j];
		}


	}
#if 1
	printf("LT = \n");

	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ", LT[i][j]);

		}
		printf("\n");
	}

#endif

	for(int i = 0; i < numVar; ++i)
		for(int j = 0; j < numVar; ++j)
		{
			M[i*numVar+j]=0;
		}

	/* Multiplying matrix L and LT and storing in M */
	for(int i = 0; i < numVar; ++i)
		for(int j = 0; j < numVar; ++j)
			for(int k = 0; k < numVar; ++k)
			{
				M[i*numVar+j] += L[i][k] * LT[k][j];

			}
#if 0
	printf("M = \n");

	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ", M[i*numVar+j]);

		}
		printf("\n");
	}

#endif




	M[numVar*numVar] = input[numVar*numVar];



	/* copy the values of M to the constant memory */

	err= hipMemcpyToSymbol(HIP_SYMBOL(MDevice),M, (numVar*numVar+1)*sizeof(float));
	//for(int i=0; i<numVar*numVar+1; i++)MDevice[i] = M[i];

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy matrix M from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	float *dataDevice;


	// allocate the memory on the GPU for the data matrix
	err = hipMalloc(&dataDevice, N *(numVar+1) * sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector data (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(dataDevice, data, N *(numVar+1) *sizeof(float), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	float *kernelValuesDevice;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&kernelValuesDevice, N*N* sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector kernel values (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	int number_of_blocks = (N*N+number_of_threads_per_block-1)/number_of_threads_per_block;
	printf("Launching the first kernel with %d blocks...\n",number_of_blocks);




	calculateKernelValues<<<number_of_blocks,number_of_threads_per_block>>>(dataDevice, kernelValuesDevice, N);
	hipDeviceSynchronize();



	printf("Kernel: calculateKernelValues is done ...\n");

#if 1

	/* this part is for validation */

	mat Mval(numVar,numVar);
	mat Xval(N,numVar);
	vec ys(N);

	float *kernelValuesHost = new float[N*N];


	err = hipMemcpy(kernelValuesHost, kernelValuesDevice, N*N*sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector kernelValues from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	for(int i=0; i<numVar; i++){

		for(int j=0; j<numVar; j++){

			Mval(i,j) = M[i*numVar+j];

		}

	}

	printf("Mval = \n");
	Mval.print();

	for(int i=0; i<N; i++){

		for(int j=0; j<numVar; j++){

			Xval(i,j) = data[i*(numVar+1)+j];

		}

		ys(i) = data[i*(numVar+1)+(numVar)];

	}
	printf("Xval = \n");
	Xval.print();

	printf("ys = \n");
	ys.print();



	float sigma  = input[numVar*numVar];
	rowvec xi,xj;

	for(int i=0; i<N; i++){

		for(int j=i+1; j<N; j++){

			xi = Xval.row(i);
			xj = Xval.row(j);

			float kernelValCPU = calcKernelValCPU(xi, xj, Mval, sigma);
			float kernelValGPU = kernelValuesHost[i*N+j];
			printf("kernelValCPU = %19.7f, kernelValGPU = %19.7f, error = %15.12f\n",kernelValCPU,kernelValGPU,kernelValCPU-kernelValGPU);


		}



	}



	delete[] kernelValuesHost;

#endif	

	/* allocate the memory on the GPU for the kernelsum */
	float *lossSumDevice;

	err = hipMalloc(&lossSumDevice, N * sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector lossSumDevice (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	float *lossSumHost = new float[N];



	number_of_blocks = (N+number_of_threads_per_block-1)/number_of_threads_per_block;
	printf("Launching the second kernel with %d blocks...\n",number_of_blocks);


	if(lossFunType == L1_LOSS_FUNCTION){

		calculateLossKernelL1<<<number_of_blocks,number_of_threads_per_block>>>(dataDevice,kernelValuesDevice, lossSumDevice, N);

	}

	hipDeviceSynchronize();



	printf("Kernel: calculateLossKernel is done ...\n");


	err = hipMemcpy(lossSumHost, lossSumDevice, N*sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector lossSum from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




#if 0

	vec lossValCPU(N);

	/* this part is for validation */
	for (int i=0; i<N; i++) {

		rowvec xi = Xval.row(i);

		float kernelSum=0.0;
		for(int j=0; j<N; j++){
			rowvec xj = Xval.row(j);

			if(i !=j){

				float kernelVal = calcKernelValCPU(xi, xj, Mval, sigma);
				kernelSum += kernelVal; 

			}


		}

		float sum = 0.0;
		for(int j=0; j<N; j++){
			rowvec xj = Xval.row(j);
			float kernelVal = calcKernelValCPU(xi, xj, Mval, sigma);
			if(i !=j){

				sum+=ys(j)*kernelVal;

			}






		}

		sum = sum/kernelSum;



		lossValCPU(i) = (ys(i)-sum)*(ys(i)-sum);

	}




	for (int i=0; i<N; i++) {
		printf( "lossGPU[%d] = %10.7f, lossGPU[%d] = %10.7f\n", i,lossSumHost[i],i, lossValCPU(i));
	}
#endif


	float totalLoss=0.0;
	for (int i=0; i<N; i++) {

		totalLoss+=lossSumHost[i];
	}


	*result = totalLoss/N;

	hipEventRecord( stop, 0 ) ;
	hipEventSynchronize( stop ) ;
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime,start, stop ) ;
	printf( "Time to generate:%3.1f ms\n", elapsedTime );


	hipEventDestroy( start ) ;
	hipEventDestroy( stop ) ;

	delete[] lossSumHost;


	hipFree(lossSumDevice);
	hipFree(kernelValuesDevice);
	hipFree(dataDevice);




}



void calcLossFunGPU_b(float *result, float *resultb, float *input,
		float *inputb, float *data, int N, int lossFunType)
{


#if 0	
	printf("calling calcLossFunGPU_b...\n");
	printf("resultb = %10.7f\n",*resultb);
	printf("Data has %d points\n",N);
#endif	


#if 0	
	hipEvent_t start, stop;
	hipEventCreate( &start ) ;
	hipEventCreate( &stop ) ;
	hipEventRecord( start, 0 ) ;
#endif

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	float LT[numVar][numVar];
	float LTb[numVar][numVar];
	float L[numVar][numVar];
	float Lb[numVar][numVar];
	float M[numVar*numVar + 1];
	float Mb[numVar*numVar + 1];


	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			L[i][j] = input[i*numVar + j];
			Lb[i][j] = 0.0;
			LT[i][j] = 0.0;
			LTb[i][j] = 0.0;

		}
	}


	for (int i = 0; i < numVar; ++i) {
		for (int j = 0; j < i+1; ++j)

			LT[j][i] = L[i][j];
	}

#if 0
	printf("L = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",L[i][j]);
		}
		printf("\n");
	}

	printf("LT = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",LT[i][j]);
		}
		printf("\n");
	}

#endif	





	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j) {

			M[i*numVar + j] = 0;
			Mb[i*numVar + j] = 0;
		}


	/* Multiplying matrix L and LT and storing in M */
	for (int i = 0; i < numVar; ++i)
		for (int j = 0; j < numVar; ++j)
			for (int k = 0; k < numVar; ++k)
				M[i*numVar + j] = M[i*numVar + j] + L[i][k]*LT[k][j];


#if 0
	printf("M = \n");
	for (int i = 0; i < numVar; ++i){
		for (int j = 0; j < numVar; ++j){

			printf("%10.7f ",M[i*numVar + j]);
		}
		printf("\n");
	}
#endif	

	M[numVar*numVar] = input[numVar*numVar];


#if 0
	printf("sigma = %10.7f\n", M[numVar*numVar]);
#endif


	/* copy the values of M to the constant memory "MDevice"*/

	err= hipMemcpyToSymbol(HIP_SYMBOL(MDevice),M, (numVar*numVar+1)*sizeof(float));
	//for(int i=0; i<numVar*numVar+1; i++)MDevice[i] = M[i];


	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy matrix M from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	float *dataDevice;


	/* allocate the memory on the GPU for the data matrix */
	err = hipMalloc(&dataDevice, N *(numVar+1) * sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector data (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(dataDevice, data, N *(numVar+1) *sizeof(float), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	float *kernelValuesDevice;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&kernelValuesDevice, N*N* sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector --kernelValuesDevice-- (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemset(kernelValuesDevice, 0, N*N* sizeof(float));

	float *kernelValuesDeviceb;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&kernelValuesDeviceb, N*N* sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector --kernelValuesDeviceb-- (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//	hipMemset(kernelValuesDeviceb, 0, N*N* sizeof(float));


	float *kernelValuesHostb = new float[N*N];

	for(int i=0; i<N*N; i++) {

		kernelValuesHostb[i] = 0.0;

	}

	err = hipMemcpy(kernelValuesDeviceb, kernelValuesHostb, (N*N) *sizeof(float), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector kernelValuesDeviceb from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}





	float *MDeviceb;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&MDeviceb, (numVar*numVar + 1)* sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector MDeviceb (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	float *MHostb = new float[numVar*numVar + 1];

	for(int i=0; i<numVar*numVar + 1; i++) {

		MHostb[i] = 0.0;

	}


	err = hipMemcpy(MDeviceb, MHostb, (numVar*numVar + 1) *sizeof(float), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector MHostb from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	delete[] MHostb;

	/* init adjoint of M to zero */
	//	hipMemset(MDeviceb, 0, (numVar*numVar + 1)* sizeof(float));







	int number_of_blocks = (N*N+number_of_threads_per_block-1)/number_of_threads_per_block;

#if 0	
	printf("Launching the first primal kernel with %d blocks...\n",number_of_blocks);
#endif

	calculateKernelValues<<<number_of_blocks,number_of_threads_per_block>>>(dataDevice, kernelValuesDevice, N);
	hipDeviceSynchronize();

#if 0
	printf("The primal kernel : calculateKernelValues is done...\n");
#endif



	float *lossSumDevice;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&lossSumDevice, N*sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector --lossSumDevice-- (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemset(lossSumDevice,0,N*sizeof(float));


	number_of_blocks = (N+number_of_threads_per_block-1)/number_of_threads_per_block;
#if 0
	printf("Launching the second primal kernel + adjoint with %d blocks...\n",number_of_blocks);
#endif


	float totalLossb = 0.0;
	totalLossb = *resultb/N;



	float *lossSumHostb = new float[N];

	for(int i=0; i<N;i++) {

		lossSumHostb[i] = 0;;
	}


	float *lossSumDeviceb;

	// allocate the memory on the GPU for kernel Values
	err = hipMalloc(&lossSumDeviceb, N*sizeof(float) ) ;

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector --lossSumDeviceb-- (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemset(lossSumDeviceb,0,N*sizeof(float));

	for (int i = N-1; i > -1; --i)
		lossSumHostb[i] = lossSumHostb[i] + totalLossb;
#if 0
	for (int i = N-1; i > -1; --i)
		printf("lossSumHostb[i] = %10.7f\n",i,lossSumHostb[i]);
#endif	

	err = hipMemcpy(lossSumDeviceb, lossSumHostb, N *sizeof(float), hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector --lossSumDevice-- from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	//	hipMemset(kernelValuesDeviceb, 0, (N*N)* sizeof(float));

	/* this subroutine evaluates the lossSumDevice and kernelValuesDeviceb */

	if(lossFunType == L1_LOSS_FUNCTION){

		calculateLossKernelL1_b<<<number_of_blocks,number_of_threads_per_block>>>(dataDevice,kernelValuesDevice,kernelValuesDeviceb, lossSumDevice,lossSumDeviceb, N);


	}

	if(lossFunType == L2_LOSS_FUNCTION){

		calculateLossKernelL2_b<<<number_of_blocks,number_of_threads_per_block>>>(dataDevice,kernelValuesDevice,kernelValuesDeviceb, lossSumDevice,lossSumDeviceb, N);


	}




	hipDeviceSynchronize();



	//hipDeviceSynchronize();
#if 0
	printf("Kernel: calculateLossKernel_b is done ...\n");
#endif


	err = hipMemcpy(kernelValuesHostb, kernelValuesDeviceb, N*N *sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector --kernelValues-- from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}




	float *lossSumHost = new float[N]();


	err = hipMemcpy(lossSumHost, lossSumDevice, N*sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector lossSum from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	float totalLoss=0.0;
	for (int i=0; i<N; i++) {

		totalLoss+=lossSumHost[i];
	}

	*result = totalLoss/N;
#if 0
	printf("result = %10.7f\n",*result);
#endif
	/* reverse sweep starts from here */



	hipMemset(MDeviceb, 0, (numVar*numVar + 1)* sizeof(float));


	number_of_blocks = (N*N+number_of_threads_per_block-1)/number_of_threads_per_block;
#if 0
	printf("Launching the second adjoint kernel with %d blocks...\n",number_of_blocks);
#endif
	/* this subroutine evaluates MDeviceb */
	calculateKernelValues_b<<<number_of_blocks,number_of_threads_per_block>>>(MDeviceb, dataDevice, kernelValuesDevice, kernelValuesDeviceb, N);



	hipDeviceSynchronize();


#if 0
	printf("Kernel: calculateKernelValues_b is done ...\n");
#endif
	for (int ii1 = 0; ii1 < numVar*numVar+1; ++ii1) {

		Mb[ii1] = 0.0;
	}

	err = hipMemcpy(Mb, MDeviceb, (numVar*numVar+1)*sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector Mb from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


#if 0
	printf("Mb = \n");

	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ", Mb[i*numVar+j]);

		}
		printf("\n");
	}



#endif	


	for (int i = numVar-1; i > -1; --i)
		for (int j = numVar-1; j > -1; --j)
			for (int k = numVar-1; k > -1; --k) {
				Lb[i][k] = Lb[i][k] + LT[k][j]*Mb[i*numVar+j];
				LTb[k][j] = LTb[k][j] + L[i][k]*Mb[i*numVar+j];
			}

	for (int i = numVar-1; i > -1; --i) {
		for (int j = i; j > -1; --j) {
			Lb[i][j] = Lb[i][j] + LTb[j][i];
			LTb[j][i] = 0.0;
		}
	}


	for (int i = numVar-1; i > -1; --i)
		for (int j = numVar-1; j > -1; --j) {
			inputb[i*numVar + j] = inputb[i*numVar + j] + Lb[i][j];
			Lb[i][j] = 0.0;
		}


	inputb[numVar*numVar] = Mb[numVar*numVar];

#if 0
	printf("inputb = \n");

	for (int i = 0; i < numVar; i++) {
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ", inputb[i*numVar+j]);

		}
		printf("\n");
	}

	printf("sigmab = %10.7f\n", inputb[numVar*numVar]);

#endif	

	hipFree(dataDevice);
	hipFree(kernelValuesDeviceb);
	hipFree(kernelValuesDevice);
	hipFree(lossSumDeviceb);
	hipFree(lossSumDevice);
	hipFree(MDeviceb);

	delete[] lossSumHost;
	delete[] lossSumHostb;
	delete[] kernelValuesHostb;

}


float kernelRegressorNotNormalized(fmat &X, fvec &y, frowvec &xp, vec& xmax, vec &xmin, fmat &M, float sigma) {

	int N = y.size();

	fvec kernelVal(N);
	fvec weight(N);

	frowvec xpNormalized;

	/* first normalize xp */

	for (int j = 0; j < xp.size(); j++) {

		xpNormalized(j) = (1.0/xp.size())*(xp(j) - xmin(j)) / (xmax(j) - xmin(j));
	}

	/* calculate the kernel values */

	float kernelSum = 0.0;
	for (int i = 0; i < N; i++) {

		frowvec xi = X.row(i);
		kernelVal(i) = gaussianKernel(xi, xpNormalized, sigma, M);
		kernelSum += kernelVal(i);
	}

	float yhat = 0.0;
	for (int i = 0; i < N; i++) {

		weight(i) = kernelVal(i) / kernelSum;
		yhat += y(i) * weight(i);
#if 0
		printf("y(%d) * weight(%d) = %10.7f * %10.7f\n",i,i,y(i),weight(i) );
#endif
	}

	return yhat;

}


/*
 * return kernel regression estimate
 * @param[in] X: sample input values (normalized)
 * @param[in] xp: point to be estimated
 * @param[in] M : Mahalanobis matrix
 * @param[in] sigma:  bandwidth parameter
 * @param[in] y: functional values at sample locations(normalized)
 *
 * */

float kernelRegressor(fmat &X, fvec &y, frowvec &xp, fmat &M, float sigma) {

	int N = y.size();

	fvec kernelVal(N);
	fvec weight(N);


	float kernelSum = 0.0;
	for (int i = 0; i < N; i++) {

		frowvec xi = X.row(i);
		kernelVal(i) = gaussianKernel(xi, xp, sigma, M);
		kernelSum += kernelVal(i);
	}



	float yhat = 0.0;
	for (int i = 0; i < N; i++) {

		weight(i) = kernelVal(i) / kernelSum;
		yhat += y(i) * weight(i);
#if 0
		printf("y(%d) * weight(%d) = %10.7f * %10.7f\n",i,i,y(i),weight(i) );
#endif
	}

	return yhat;

}





/*
 * return kernel regression estimate with gradient data
 * @param[in] X: sample input values (normalized)
 * @param[in] grad: sample gradient values (normalized)
 * @param[in] xp: point to be estimated
 * @param[in] M : Mahalanobis matrix
 * @param[in] sigma:  bandwidth parameter
 * @param[in] y: functional values at sample locations(normalized)
 *
 * */


float kernelRegressor(fmat &X, fvec &y, fmat &grad, frowvec &xp, fmat &M, float sigma) {

	int N = y.size();

	fvec kernelVal(N);
	fvec weight(N);
	float kernelSum = 0.0;
	float yhat = 0.0;




	/* first evaluate the kernel sum */
	for (int i = 0; i < N; i++) {

		frowvec xi = X.row(i);
		kernelVal(i) = gaussianKernel(xi, xp, sigma, M);
		kernelSum += kernelVal(i);


	}

#if 1

	int above99flag=0;
	int above99index = -1;
	for (int i = 0; i < N; i++) {

		weight(i) = kernelVal(i) / kernelSum;

		if(weight(i) > 0.5) {

			above99index = i;
			above99flag=1;
		}

	}

	if(above99flag == 1){
		printf("above99flag is 1 with index = %d!\n",above99index);
		for (int i = 0; i < N; i++) {

			if(i!=above99index) {

				weight(i) = 0.0;
			}
			else weight(i) = 1.0;
		}


	}


	//	weight.print();

#endif

	frowvec xdiff(xp.size());

	for (int i = 0; i < N; i++) {

		frowvec xi = X.row(i);

		for(int j=0; j<xp.size(); j++) xdiff(j) = xp(j) -xi(j);

		xdiff.print();

#if 0
		printf("xp =\n");
		xp.print();
		printf("xi =\n");
		xi.print();
		printf("xdiff =\n");
		xdiff.print();
		printf("grad =\n");
		grad.row(i).print();
#endif


		float gradTerm = dot(xdiff,grad.row(i));
#if 0
		printf("gradTerm = %10.7f\n",gradTerm);
		printf("y = %10.7f\n",y(i));
#endif



		//		weight(i) = kernelVal(i) / kernelSum;
		yhat += (y(i) + gradTerm) * weight(i);
		//		yhat += (y(i) ) * weight(i);
#if 0
		printf("y(%d) * weight(%d) = %10.7f * %10.7f\n",i,i,y(i),weight(i) );
#endif
	}


	printf("yhat = %10.7f\n",yhat);
	return yhat;

}


/*
 * return kernel regression estimate with gradient data
 * @param[in] X: sample input values (normalized)
 * @param[in] XnotNormalized: sample input values
 * @param[in] grad: sample gradient values (not normalized)
 * @param[in] xp: point to be estimated
 * @param[in] M : Mahalanobis matrix
 * @param[in] sigma:  bandwidth parameter
 * @param[in] y: functional values at sample locations
 *
 * */


float kernelRegressorNotNormalized(fmat &X,
		fmat &XnotNormalized,
		fvec &y,
		fmat &grad,
		frowvec &xp,
		fvec &xmin,
		fvec &xmax,
		fmat &M,
		float sigma) {


	/* number of samples */
	int N = y.size();
	int d = xp.size();

	fvec kernelVal(N);
	fvec weight(N);

	frowvec xpNormalized(d);

	/* first normalize xp */

	for (int j = 0; j < d; j++) {

		xpNormalized(j) = (1.0/d)*(xp(j) - xmin(j)) / (xmax(j) - xmin(j));
	}


	float kernelSum = 0.0;


	frowvec xi(d);
	frowvec xdiff(d);

	/* first evaluate the kernel sum */
	for (int i = 0; i < N; i++) {

		xi = X.row(i);

		kernelVal(i) = gaussianKernel(xi, xpNormalized, sigma, M);
		kernelSum += kernelVal(i);
	}



	float yhat = 0.0;

	for (int i = 0; i < N; i++) {


		xi = XnotNormalized.row(i);
		for(int j=0; j<d; j++) {

			xdiff(j) = xp(j) -xi(j);
		}


		float gradTerm = dot(xdiff,grad.row(i));

		weight(i) = kernelVal(i) / kernelSum;
		yhat += (y(i) + gradTerm) * weight(i);
#if 0
		printf("y(%d) * weight(%d) = %10.7f * %10.7f\n",i,i,y(i),weight(i) );
#endif
	}

	return yhat;

}


/*
 * return kernel regression estimate with gradient data
 * @param[in] X: sample input values (normalized)
 * @param[in] XnotNormalized: sample input values
 * @param[in] grad: sample gradient values (not normalized)
 * @param[in] xp: point to be estimated
 * @param[in] M : Mahalanobis matrix
 * @param[in] sigma:  bandwidth parameter
 * @param[in] y: functional values at sample locations
 *
 * */


double kernelRegressorNotNormalized(mat &X,
		mat &XnotNormalized,
		vec &y,
		mat &grad,
		rowvec &xp,
		vec &xmin,
		vec &xmax,
		mat &M,
		double sigma) {


	/* number of samples */
	int N = y.size();
	int d = xp.size();

	vec kernelVal(N);
	vec weight(N);

	rowvec xpNormalized(d);

	/* first normalize xp */

	for (int j = 0; j < d; j++) {

		xpNormalized(j) = (1.0/d)*(xp(j) - xmin(j)) / (xmax(j) - xmin(j));
	}


	double kernelSum = 0.0;


	rowvec xi(d);
	rowvec xdiff(d);

	/* first evaluate the kernel sum */
	for (int i = 0; i < N; i++) {

		xi = X.row(i);

		kernelVal(i) = gaussianKernel(xi, xpNormalized, sigma, M);
		kernelSum += kernelVal(i);
	}



	double yhat = 0.0;

	for (int i = 0; i < N; i++) {


		xi = XnotNormalized.row(i);
		for(int j=0; j<d; j++) {

			xdiff(j) = xp(j) -xi(j);
		}


		double gradTerm = dot(xdiff,grad.row(i));

		weight(i) = kernelVal(i) / kernelSum;
		yhat += (y(i) + gradTerm) * weight(i);
#if 0
		printf("y(%d) * weight(%d) = %10.7f * %10.7f\n",i,i,y(i),weight(i) );
#endif
	}

	return yhat;

}




/*
 * train the Mahalanobis matrix M and bandwidth parameter sigma
 * @param[in] data: sample data matrix (normalized values)
 * @param[in] max_cv_iter: number of iterations for cross validation loop
 * @param[out] wSvd: weight for svd regularization
 * @param[out] w12:  weight for mixed 12norm regularization
 * @param[out] M: Mahalanobis matrix
 * @param[out] sigma: bandwidth parameter for the Gaussian kernel
 *
 * */


int trainMahalanobisDistance(fmat &L, fmat &data, float &sigma, float &wSvd, float &w12,int max_cv_iter, int lossFunType) {


	bool trainWithSvdFlag = false;

	if(wSvd > 0.0) trainWithSvdFlag = true;


	int max_opt_iter = 40000;

	unsigned int n = L.n_cols;
	unsigned int m = L.n_cols;
	float alpha = 0.9;

	if(m != n || m!=numVar || n!=numVar){

		fprintf(stderr,"Cols: %d and Rows: %d\n",n, m);
		fprintf(stderr,"Error: The Mahalanobis matrix is not square!\n");
		exit(-1);
	}

	int Ldim = numVar*numVar;

	/* lower diagonal matrix Lbest to keep the best L*/
	fmat bestL(numVar,numVar);
	bestL.fill(0.0);

	float bestsigma = 0.0;


	/* divide the data set into training and validation sets */

	unsigned int N = data.n_rows;


	/* size of the validation set, default to one fifth */
	unsigned int NvalidationSet = N/5;
	unsigned int Ntraining = N - NvalidationSet;


#if 1

	printf("number of training samples (core) = %d\n",Ntraining);
	printf("number of validation samples      = %d\n",NvalidationSet);

#endif




	fmat dataTraining      = data.submat( 0, 0, Ntraining-1, numVar );
	fmat dataValidation    = data.submat( Ntraining, 0, N-1, numVar );


	fmat XValidation = dataValidation.submat(0,0,NvalidationSet-1,numVar-1);
	fvec yValidation = dataValidation.col(numVar);
	fmat XTraining = dataTraining.submat(0,0,Ntraining-1,numVar-1);
	fvec yTraining = dataTraining.col(numVar);




#if 0

	printf("Training data set = \n");
	dataTraining.print();

	printf("Validation data set = \n");
	dataValidation.print();
#endif


#if 0
	printf("XTraining = \n");
	XTraining.print();
	printf("yTraining = \n");
	yTraining.print();
#endif

#if 0
	printf("XValidation = \n");
	XValidation.print();
	printf("yValidation = \n");
	yValidation.print();
#endif

	fvec wSvdtrial(max_cv_iter);
	fvec w12trial(max_cv_iter);


	if(max_cv_iter !=1){



		for(int i=0; i<max_cv_iter; i++){

			wSvdtrial(i) = pow(10.0,RandomFloat(-2,0.0));
			w12trial(i) = pow(10.0,RandomFloat(-5,0.0));
		}


#if 1
		printf("wSvdtrial = \n");
		wSvdtrial.print();
		printf("w12trial = \n");
		w12trial.print();
#endif


	}


	float *inputVec = new float[Ldim+1]();
	float *inputVecVel = new float[Ldim+1]();
	float *inputVecLocalBest = new float[Ldim+1]();
	float *inputVecb = new float[Ldim+1]();
	float *inputVecRegb = new float[Ldim]();
	float *gradientVec = new float[Ldim+1]();
	float *dataVecTraining = new float[Ntraining*(n+1)]();




#if 0
	printf("L = \n");
	for (int i = 0; i < numVar; i++){
		for (int j = 0; j < numVar; j++) {

			printf("%10.7f ",inputVec[i*numVar+j]);
		}
		printf("\n");
	}

	printf("sigma = %10.7f\n",inputVec[Ldim]);
#endif	


#if 1
	printf("copying training data...\n");
#endif	
	for (int i = 0; i < Ntraining; i++) {

		for (int j = 0; j < numVar+1; j++) {

			dataVecTraining[i*(n+1)+j ] = dataTraining(i, j);
		}
	}
#if 0
	printf("data copied = \n");

	for (int i = 0; i < Ntraining; i++) {

		for (int j = 0; j < numVar+1; j++) {

			printf("%10.7f ",dataVecTraining[i*(n+1)+j ]);
		}
		printf("\n");
	}

#endif	


	float optGenError = 10E14;

	/* cross validation loop to tune the weights for the regularization parameters */
	for(int iter_cv=0; iter_cv< max_cv_iter; iter_cv++){


		float learning_rateM = 0.0001;
		float learning_rateSigma = learning_rateM * 0.01;



		if(max_cv_iter !=1){

			if(trainWithSvdFlag){
				wSvd = wSvdtrial(iter_cv);
			}
			else{

				wSvd = 0.0;
			}
			w12 =  w12trial(iter_cv);
		}
#if 1
		printf("Outer iteration = %d\n",iter_cv);
		printf("wSvd = %10.7f, w12 = %10.7f\n",wSvd,w12);
#endif		

		/* initialize the L matrix and sigma => everything is saved in the vector "inputVec" */

		for (int i = 0; i < numVar; i++)
			for (int j = 0; j < numVar; j++) {

				inputVec[i*numVar+j] = 0.0;
			}

		for (int i = 0; i < numVar; i++) {

			for (int j = 0; j <= i; j++) {

				if(i ==j) { /* main diagonal */

					inputVec[i*numVar+j] = 1.0+ RandomFloat(-0.1,0.1);
				}
				else {

					inputVec[i*numVar+j] = RandomFloat(0.0,0.1);
				}
			}
		}

		/* assign sigma */
		inputVec[Ldim] = RandomFloat(0.0,0.1);

		float lossVal,lossValb, regTerm;
		float objFunVal;
		lossVal = 0.0;
		lossValb = 1.0;

		for(int i=0;i<Ldim+1;i++) {

			inputVecb[i] = 0.0;
		}

		/* calculate the first gradient vector */

		printf("Evaluating the first gradient...\n");

		calcLossFunGPU_b(&lossVal, &lossValb, inputVec,inputVecb, dataVecTraining,Ntraining, lossFunType);

		printf("initial Loss (GPU Version)= %10.7f\n", lossVal);


#if 0
		printf("gradient of the loss term = \n");

		for (int i = 0; i < numVar; i++) {
			for (int j = 0; j < numVar; j++) {

				printf("%10.7f ", inputVecb[i*numVar+j]);

			}
			printf("\n");
		}
		printf("sigma sensitivity = %10.7f\n", inputVecb[Ldim]);
#endif





		for(int i=0;i<Ldim+1;i++) {

			gradientVec[i]=inputVecb[i];
		}


#if 0

		/* call the CodiPack version for validation */

		codi::RealReverse *inputVecCodi = new codi::RealReverse[n*n+1];


		for(int i=0; i<n*n+1; i++){

			inputVecCodi[i] = inputVec[i];

		}

		codi::RealReverse lossValCodi = 0.0;
		float *inputVecbCodi = new float[n*n+1]();

		/* call the CodiPack version of "calcLossFunCPU" */ 

		printf("calling calcLossFunCPU (reverse AD)...\n");
		calcLossFunCPU(&lossValCodi,inputVecCodi, inputVecbCodi, dataVecTraining, Ntraining);

		printf("Lb (codipack result)= \n");

		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {

				printf("%10.7f ", inputVecbCodi[i*n+j]);

			}
			printf("\n");
		}
		printf("sigmab = %10.7f\n", inputVecbCodi[n*n]);
		printf("lossValCodi = %10.7f\n", lossValCodi.getValue());	


#endif



#if 0
		printf("calculating regularization term...\n");
#endif
		for(int i=0;i<Ldim;i++) {

			inputVecRegb[i] = 0.0;
		}

		/* call the adjoint mode of the function to compute the regularization term */
		if(trainWithSvdFlag){

			calcRegTerms(inputVec, inputVecRegb, &regTerm, wSvd, w12, n);
		}
		else{

			calcRegTermL12(inputVec, inputVecRegb,&regTerm, w12, n);

		}

#if 0	
		printf("gradient of the regularization term = \n");

		for (int i = 0; i < numVar; i++) {
			for (int j = 0; j < numVar; j++) {

				printf("%10.7f ", inputVecRegb[i*numVar+j]);

			}
			printf("\n");
		}
#endif

		objFunVal = lossVal + regTerm;

		printf("initial value of the objective function = %10.7f\n",objFunVal);

		/* add the regularization sensitivities to the gradient vector */

		for(int i=0;i<Ldim;i++) {

			gradientVec[i]+=inputVecRegb[i];
		}


#if 0	

		/* validation loop for the regularization term */


		float f0 = 0.0;
		float tempSave;
		calcRegTerms(inputVec, &f0, wSvd, w12, n);
		printf("f0 = %10.7f\n",f0);
		float epsValReg= 0.001;


		for (int i = 0; i < n; i++) {
			for (int j = 0; j <= i; j++) {

				printf("validating the (%d,%d) th element of M\n",i,j);
				tempSave = inputVec[i*n+j];
				inputVec[i*n+j]+=epsValReg;

				float f1 = 0.0;

				calcRegTerms(inputVec, &f1, wSvd, w12, n);
				printf("f1 = %10.7f, f0 = %10.7f\n",f1,f0);
				inputVec[i*n+j]= tempSave;

				float fdVal = (f1-f0)/epsValReg;

				printf("fd value = %10.7f, ad value = %10.7f\n",fdVal,inputVecRegb[i*n+j]);

				float f2,f2d;

				/* call forward mode */
				calcRegTerms(inputVec, &f2,&f2d, wSvd, w12, n, i*n+j);

				printf("primal value = %10.7f, forward ad value = %10.7f, ad value = %10.7f\n",f2,f2d,inputVecRegb[i*n+j]);


			}

		}




#endif

		/* optimization loop */


		/* check gradient */
		for(int i=0;i<Ldim;i++) {

			if( gradientVec[i] != gradientVec[i]){

				printf("gradientVec[%d] is NaN!\n",i);
				exit(1);

			}
		}


		float objectiveFunLocalBest = 10E14;



		for(int opt_iter=0 ; opt_iter < max_opt_iter; opt_iter++){



			/* update M */

			for (int i = 0; i < numVar; i++){
				for (int j = 0; j <= i; j++) {

					inputVec[i*numVar+j]= inputVec[i*numVar+j] + inputVecVel[i*numVar+j];


				}

			}


			for (int i = 0; i < numVar; i++){
				for (int j = 0; j <= i; j++) {

					if ( inputVec[i*numVar+j] < 0) {

						inputVec[i*numVar+j] = 10E-6;

					}


				}

			}

			/* update sigma */
			inputVec[Ldim]= inputVec[Ldim] + inputVecVel[Ldim];

			if(inputVec[Ldim] <= 0) {

				inputVec[Ldim] = 10E-06;
			}


			for(int i=0;i<Ldim+1;i++) {

				inputVecb[i] = 0.0;
			}

			/* calculate the gradient vector */
#if 0
			printf("evaluating gradient vector...\n");
#endif		

			calcLossFunGPU_b(&lossVal, &lossValb, inputVec,inputVecb, dataVecTraining,Ntraining,L1_LOSS_FUNCTION);


#if 0
			printf("Loss (GPU Version)= %10.7f\n", lossVal);
#endif
			for(int i=0;i<Ldim+1;i++) {

				gradientVec[i]=inputVecb[i];
			}
#if 0
			printf("calculating the regularization term...\n");
#endif
			for(int i=0;i<Ldim;i++) {

				inputVecRegb[i] = 0.0;
			}

			/* call the adjoint mode of the function to compute the regularization term */

			if(trainWithSvdFlag){

						calcRegTerms(inputVec, inputVecRegb, &regTerm, wSvd, w12, n);
					}
					else{

						calcRegTermL12(inputVec, inputVecRegb,&regTerm, w12, n);

					}


#if 0	
			printf("gradient of the regularization term = \n");

			for (int i = 0; i < numVar; i++) {
				for (int j = 0; j < numVar; j++) {

					printf("%10.7f ", inputVecRegb[i*numVar+j]);

				}
				printf("\n");
			}
#endif		


			/* add the regularization sensitivities to the gradient vector */

			for(int i=0;i<Ldim;i++) {

				gradientVec[i]+=inputVecRegb[i];
			}


			objFunVal = lossVal + regTerm;

			if(objFunVal < objectiveFunLocalBest){

				objectiveFunLocalBest = objFunVal;

				for(int i=0;i<Ldim+1;i++) {

					inputVecLocalBest[i]=inputVec[i];

				}




			}





			if(opt_iter % 100 == 0){


				printf("iter = %d, objective function = %10.7f, Leave One Out Error = %10.7f, Regularization term = %10.7f\n",opt_iter,objFunVal,lossVal, regTerm);

#if 0
				printf("L = \n");

				for (int i = 0; i < numVar; i++) {
					for (int j = 0; j < numVar; j++) {

						printf("%10.7f ", inputVec[i*numVar+j]);

					}
					printf("\n");
				}

				printf("sigma = %10.7f\n",inputVec[Ldim]);
#endif



			}


			/* update velocity vector */
			for(int i=0;i<Ldim;i++) {

				inputVecVel[i]=alpha* inputVecVel[i] - learning_rateM*gradientVec[i];

			}
			inputVecVel[Ldim]=alpha* inputVecVel[Ldim] - learning_rateSigma*gradientVec[Ldim];




		} /* end of local optimization loop */



		for (int i = 0; i < numVar; i++)
			for (int j = 0; j < numVar; j++) {

				L(i,j)= inputVecLocalBest[i*numVar+j];
			}

#if 1
		printf("local optimization result:\n");
		printf("L = \n");
		L.print();
		printf("sigma = %10.7f\n", inputVecLocalBest[Ldim]);

#endif
		sigma = inputVecLocalBest[Ldim];


		fmat M = L*trans(L);
#if 1
		printf("M = \n");
		M.print();
#endif
		float genError = 0.0;

		for(int i=0;i <NvalidationSet; i++){

			frowvec xp = XValidation.row(i);
			float ytilde = kernelRegressor(XTraining, yTraining, xp, M, sigma);
			float yexact = yValidation(i);

#if 0
			printf("x:\n");
			xp.print();
			printf("ytilde = %10.7f, yexact = %10.7f\n",ytilde,yexact);
#endif



			if( lossFunType == L1_LOSS_FUNCTION) genError += fabs(yexact-ytilde);
			if( lossFunType == L2_LOSS_FUNCTION) genError += (yexact-ytilde)*(yexact-ytilde);

		}

		genError = genError/NvalidationSet;

#if 1
		printf("Generalization error = %10.7f\n",genError);
#endif		
		if(genError < optGenError) {

#if 1
			printf("Better L has been found, updating L...\n");
#endif			
			bestL = L;
			bestsigma = sigma;
			optGenError = genError;


		}



	} /* end of cv loop */

	L = bestL;
	sigma = bestsigma;


	delete[] inputVec;
	delete[] inputVecb;
	delete[] inputVecRegb;
	delete[] dataVecTraining;
	delete[] gradientVec;
	return 0;

}




